#include "hip/hip_runtime.h"
//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "include\_simulationvariables.h" //didn't add to this vs project - each project this class is attached to will have its own variables header
#include "SatelliteClass\Satellite.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error %d at %s:%d\n",EXIT_FAILURE,__FILE__,__LINE__);}} while(0)

__global__ void setupKernel(double* array1D, double** array2D, int cols, int entrs)
{
	if (blockIdx.x * blockDim.x + threadIdx.x != 0)
		return;

	for (int iii = 0; iii < cols; iii++)
		array2D[iii] = &array1D[iii * entrs];
}

__global__ void satelliteDetector(double** data_d, double** capture_d, double simtime, double altitude, bool upward)
{
	int thdInd = blockIdx.x * blockDim.x + threadIdx.x;

	double* v_d; double* mu_d; double* z_d; double* simtime_d;
	double* detected_v_d; double* detected_mu_d; double* detected_z_d;
	v_d = data_d[0]; mu_d = data_d[1]; z_d = data_d[2]; simtime_d = capture_d[3];
	detected_v_d = capture_d[0]; detected_mu_d = capture_d[1]; detected_z_d = capture_d[2];

	double z_minus_vdt{ z_d[thdInd] - v_d[thdInd] * DT };
	
	if (simtime == 0) //not sure I fully like this, but it works
		simtime_d[thdInd] = -1.0;

	if (
		(detected_z_d[thdInd] < 1) && ( //no detected particle is in the data array at the thread's index already AND
		//detector is facing down and particle crosses altitude in dt
		((!upward) && (z_d[thdInd] > altitude) && (z_minus_vdt < altitude))
		|| //OR
		//detector is facing up and particle crosses altitude in dt
		((upward) && (z_d[thdInd] < altitude) && (z_minus_vdt > altitude)) ) 
		)
	{
		detected_v_d[thdInd] = v_d[thdInd];
		detected_mu_d[thdInd] = mu_d[thdInd];
		detected_z_d[thdInd] = z_d[thdInd];
		simtime_d[thdInd] = simtime;
	}//particle not removed from sim
}

void Satellite::initializeSatelliteOnGPU()
{
	CUDA_CALL(hipMalloc((void **)&satCaptureGPU_m, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m)); //makes room for data of detected particles
	CUDA_CALL(hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m)); //sets values to 0
	CUDA_CALL(hipMalloc((void **)&dblppGPU_m[1], sizeof(double*) * numberOfAttributes_m));

	setupKernel <<< 1, 1 >>> (satCaptureGPU_m, dblppGPU_m[1], numberOfAttributes_m + 1, numberOfParticles_m);
}

void Satellite::iterateDetector(int numberOfBlocks, int blockSize, double simtime) {
	satelliteDetector <<< numberOfBlocks, blockSize >>>	(dblppGPU_m.at(0), dblppGPU_m.at(1), simtime, altitude_m, upwardFacing_m); }

void Satellite::copyDataToHost()
{// data_m array: [v_para, mu, z, time][particle number]
	for (int satattr = 0; satattr < numberOfAttributes_m + 1; satattr++)
		CUDA_CALL(hipMemcpy(data_m.at(satattr).data(), satCaptureGPU_m + satattr * numberOfParticles_m, sizeof(double) * numberOfParticles_m, hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m)); //sets values to 0

	dataReady_m = true;
}

void Satellite::freeGPUMemory()
{
	CUDA_CALL(hipFree(satCaptureGPU_m));
	CUDA_CALL(hipFree(dblppGPU_m.at(1))); //DO NOT FREE dblppGPU_m[0] - this is the 2D data array that the sim uses (not the satellite)
}

void Satellite::vectorTest(std::vector<double*>& in)
{
	int wrong{ 0 };
	for (int iii = 0; iii < numberOfAttributes_m; iii++)
		for (int jjj = 0; jjj < numberOfParticles_m; jjj++)
			if (in.at(iii)[jjj] != data_m.at(iii)[jjj]) { wrong++; }

	std::cout << "Wrong: " << wrong << "\n";
}