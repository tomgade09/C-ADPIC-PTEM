#include "hip/hip_runtime.h"
//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "include\_simulationvariables.h" //didn't add to this vs project - each project this class is attached to will have its own variables header
#include "SatelliteClass\Satellite.h"

__global__ void setupKernel(double* array1D, double** array2D, int cols, int entrs)
{
	if (blockIdx.x * blockDim.x + threadIdx.x != 0)
		return;

	for (int iii = 0; iii < cols; iii++)
		array2D[iii] = &array1D[iii * entrs];
}

__global__ void satelliteDetector(double** data_d, double** capture_d, double simtime, double altitude, bool upward)
{
	int thdInd = blockIdx.x * blockDim.x + threadIdx.x;

	double* v_d; double* mu_d; double* z_d; double* simtime_d;
	double* detected_v_d; double* detected_mu_d; double* detected_z_d;
	v_d = data_d[0]; mu_d = data_d[1]; z_d = data_d[2]; simtime_d = capture_d[3];
	detected_v_d = capture_d[0]; detected_mu_d = capture_d[1]; detected_z_d = capture_d[2];

	double z_minus_vdt{ z_d[thdInd] - v_d[thdInd] * DT };
	
	if (simtime == 0) //not sure I fully like this, but it works
		simtime_d[thdInd] = -1.0;

	if (
		(detected_z_d[thdInd] < 1) && ( //no detected particle is in the data array at the thread's index already AND
		//detector is facing down and particle crosses altitude in dt
		((!upward) && (z_d[thdInd] > altitude) && (z_minus_vdt < altitude))
		|| //OR
		//detector is facing up and particle crosses altitude in dt
		((upward) && (z_d[thdInd] < altitude) && (z_minus_vdt > altitude)) ) 
		)
	{
		detected_v_d[thdInd] = v_d[thdInd];
		detected_mu_d[thdInd] = mu_d[thdInd];
		detected_z_d[thdInd] = z_d[thdInd];
		simtime_d[thdInd] = simtime;
	}//particle not removed from sim
}

void Satellite::initializeSatelliteOnGPU()
{
	hipMalloc((void **)&satCaptureGPU_m, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m); //makes room for data of detected particles
	hipMemset((void **)&satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m); //sets values to 0
	hipMalloc((void **)&dblppGPU_m[1], sizeof(double*) * numberOfAttributes_m);

	setupKernel <<< 1, 1 >>> (satCaptureGPU_m, dblppGPU_m[1], numberOfAttributes_m + 1, numberOfParticles_m);
}

void Satellite::iterateDetector(int numberOfBlocks, int blockSize, double simtime) {
	satelliteDetector <<< numberOfBlocks, blockSize >>>	(dblppGPU_m[0], dblppGPU_m[1], simtime, altitude_m, upwardFacing_m); }

void Satellite::copyDataToHost()
{// data_m array: [v_para, mu, z, time][particle number]
	hipMemcpy(data_m[0], satCaptureGPU_m, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m, hipMemcpyDeviceToHost);
	hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 1) * numberOfParticles_m); //sets values to 0

	dataReady_m = true;
}

void Satellite::freeGPUMemory()
{
	hipFree(satCaptureGPU_m);
	hipFree(dblppGPU_m[1]); //DO NOT FREE dblppGPU_m[0] - this is the 2D data array that the sim uses (not the satellite)
}

void Satellite::vectorTest(std::vector<double*>& in)
{
	int wrong{ 0 };
	for (int iii = 0; iii < numberOfAttributes_m; iii++)
		for (int jjj = 0; jjj < numberOfParticles_m; jjj++)
			if (in[iii][jjj] != data_m[iii][jjj]) { wrong++; }

	std::cout << "Wrong: " << wrong << "\n";
}