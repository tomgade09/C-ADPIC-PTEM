#include "hip/hip_runtime.h"
#include "include\AlfvenLUT.h"

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error %d at %s:%d\n",EXIT_FAILURE,__FILE__,__LINE__);}} while(0)
__global__ void setup2DArray(double* array1D, double** array2D, int cols, int entries);

extern const int SIMCHARSIZE;

__host__ __device__ double alfvenWaveEbyLUT(double** LUT, double z, double simtime, double omegaE)
{//E Field in the direction of B (radially outward)
	if (LUT == nullptr)
		return 0.0;
	if (z > RADIUS_EARTH) //in case z is passed in as m, not Re, convert to Re
		z = z / RADIUS_EARTH;
	if (z < LUT[0][0] || z > LUT[0][2950])
		return 0.0;

	double offset{ LUT[0][0] };
	int stepsFromZeroInd{ static_cast<int>(floor((z - offset) / (LUT[0][1] - LUT[0][0]))) }; //only works for constant bin size - if the binsize changes throughout LUT, need to iterate which will take longer

	//y = mx + b
	double linearInterpReal{ ((LUT[1][stepsFromZeroInd + 1] - LUT[1][stepsFromZeroInd]) / (LUT[0][stepsFromZeroInd + 1] - LUT[0][stepsFromZeroInd])) *
		(z - LUT[0][stepsFromZeroInd]) + LUT[1][stepsFromZeroInd] };
	double linearInterpImag{ ((LUT[2][stepsFromZeroInd + 1] - LUT[2][stepsFromZeroInd]) / (LUT[0][stepsFromZeroInd + 1] - LUT[0][stepsFromZeroInd])) *
		(z - LUT[0][stepsFromZeroInd]) + LUT[2][stepsFromZeroInd] };

	//E-par = (column 2)*cos(omega*t) + (column 3)*sin(omega*t), omega - angular frequency of wave
	return (linearInterpReal * cos(omegaE * simtime) + linearInterpImag * sin(omegaE * simtime)) / 1000; //LUT E is in mV / m
}

void AlfvenLUT::initializeFollowOn()
{
	useAlfLUT_m = true;
	CUDA_CALL(hipMalloc((void **)&gpuDblMemoryPointers_m.at(2 * particleTypes_m.size() + 1), numOfColsLUT_m * numOfEntrLUT_m * sizeof(double)));
	CUDA_CALL(hipMalloc((void **)&gpuOtherMemoryPointers_m.at(2 * particleTypes_m.size() + 1), numOfColsLUT_m * sizeof(double*)));
}

void AlfvenLUT::copyDataToGPUFollowOn()
{
	CUDA_CALL(hipMemcpy(gpuDblMemoryPointers_m.at(2 * particleTypes_m.size() + 1), elcFieldLUT_m[0], numOfColsLUT_m * numOfEntrLUT_m * sizeof(double), hipMemcpyHostToDevice));
	setup2DArray <<< 1, 1 >>> (gpuDblMemoryPointers_m.at(2 * particleTypes_m.size() + 1), reinterpret_cast<double**>(gpuOtherMemoryPointers_m.at(2 * particleTypes_m.size() + 1)), numOfColsLUT_m, numOfEntrLUT_m);
	CUDA_CALL(hipMemcpy(gpuDblMemoryPointers_m.at(2 * particleTypes_m.size()) + (SIMCHARSIZE/sizeof(double) - 1), &omegaE_m, sizeof(double), hipMemcpyHostToDevice));
}

void AlfvenLUT::iterateSimulationFollowOnPreLoop()
{
	if (useAlfLUT_m && gpuOtherMemoryPointers_m.at(2 * particleTypes_m.size() + 1) == nullptr)
		std::cout << "Warning: LUT pointer is a nullptr.  Alfven wave function will return 0.0.  Continuing.\n";
}

void AlfvenLUT::iterateSimulationFollowOnInsideLoop()
{
	return;
}

void AlfvenLUT::iterateSimulationFollowOnPostLoop()
{
	return;
}

void AlfvenLUT::copyDataToHostFollowOn()
{
	return;
}

void AlfvenLUT::freeGPUMemoryFollowOn()
{
	return;
}