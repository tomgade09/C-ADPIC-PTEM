#include "hip/hip_runtime.h"
//Standard Library includes
#include <string>
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <time.h>
#include <vector>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "include\_simulationvariables.h"
#include "include\Simulation170925.h"

//Array Size Variables
const int DBLARRAY_BYTES { NUMPARTICLES * sizeof(double) }; //Global vars - not my favorite solution, but I suppose it works for now
const int BOOLARRAY_BYTES{ NUMPARTICLES * sizeof(bool) };
const int INTARRAY_BYTES { NUMPARTICLES * sizeof(int) };

//Angular Freq of Efield
constexpr double OMEGA{ 2 * PI / 10 };

__host__ double EFieldatZ(double** LUT, double z, double simtime)
{//E Field in the direction of B (radially outward)
	//E-par = (column 2)*cos(omega*t) + (column 3)*sin(omega*t), omega = 2 PI / 10
	if (z > RADIUS_EARTH) //in case z is passed in as m, not Re
		z = z / RADIUS_EARTH; //convert to Re
	
	if (z < LUT[0][0] || z > LUT[0][2950])
		return 0.0;
	double offset{ LUT[0][0] };
	int stepsFromZeroInd{ static_cast<int>(floor((z - offset) / (LUT[0][1] - LUT[0][0]))) }; //only works for constant bin size - if the binsize changes throughout LUT, need to iterate which will take longer
	
	//y = mx + b
	double linearInterpReal{ ((LUT[1][stepsFromZeroInd + 1] - LUT[1][stepsFromZeroInd]) / (LUT[0][stepsFromZeroInd + 1] - LUT[0][stepsFromZeroInd])) *
		(z - LUT[0][stepsFromZeroInd]) + LUT[1][stepsFromZeroInd] };
	double linearInterpImag{ ((LUT[2][stepsFromZeroInd + 1] - LUT[2][stepsFromZeroInd]) / (LUT[0][stepsFromZeroInd + 1] - LUT[0][stepsFromZeroInd])) *
		(z - LUT[0][stepsFromZeroInd]) + LUT[2][stepsFromZeroInd] };
	
	return linearInterpReal * cos(OMEGA * simtime) + linearInterpImag * sin(OMEGA * simtime);
}

__device__ double EFieldatZ(double* LUT, double z, double simtime)//biggest concern here
{//E Field in the direction of B (radially outward)
	if (z > RADIUS_EARTH) //in case z is passed in as m, not Re
		z = z / RADIUS_EARTH; //convert to Re
	
	if (z < LUT[0] || z > LUT[2950])
		return 0.0;
	double offset{ LUT[0] };
	int stepsFromZeroInd{ static_cast<int>(floor((z - offset) / (LUT[1] - LUT[0]))) }; //only works for constant bin size - if the binsize changes throughout LUT, need to iterate which will take longer
	
	//y = mx + b
	double linearInterpReal{ ((LUT[2951 + stepsFromZeroInd + 1] - LUT[2951 + stepsFromZeroInd]) / (LUT[stepsFromZeroInd + 1] - LUT[stepsFromZeroInd])) * 
		(z - LUT[stepsFromZeroInd]) + LUT[2951 + stepsFromZeroInd] };
	double linearInterpImag{ ((LUT[2 * 2951 + stepsFromZeroInd + 1] - LUT[2 * 2951 + stepsFromZeroInd]) / (LUT[stepsFromZeroInd + 1] - LUT[stepsFromZeroInd])) * 
		(z - LUT[stepsFromZeroInd]) + LUT[2 * 2951 + stepsFromZeroInd] };

	return linearInterpReal * cos(OMEGA * simtime) + linearInterpImag * sin(OMEGA * simtime);
}

__host__ __device__ double BFieldatZ(double z, double simtime) //this will change in future iterations
{//for now, a simple dipole field
	return DIPOLECONST * pow((RADIUS_EARTH / NORMFACTOR) / z, 3); //Bz = B0 * (r0/rz)^3, r0=DIPOLECONST - B at 1 Re
}

__global__ void initCurand(hiprandStateMRG32k3a* state, long long seed)
{
	long long id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__device__ double normalGeneratorCUDA(hiprandStateMRG32k3a* state, long long id, double mean, double sigma)
{
	hiprandStateMRG32k3a localState = state[id];
	
	double res = sigma * hiprand_normal_double(&localState) + mean;
	state[id] = localState;

	return res;
}

__device__ double accel1dCUDA(double* args, int len, double* LUT) //made to pass into 1D Fourth Order Runge Kutta code
{//args array: [t_RK, vz, mu, q, m, pz_0, simtime]
	double F_lor, F_mir, ztmp;
	ztmp = args[5] + args[1] * args[0]; //pz_0 + vz * t_RK
	
	//Lorentz force - simply qE - v x B is taken care of by mu - results in kg.m/s^2 - to convert to Re equivalent - divide by Re
	F_lor = args[3] * EFieldatZ(LUT, ztmp, args[6] + args[0]) / NORMFACTOR; //will need to replace E with a function to calculate in more complex models

	//Mirror force
	F_mir = -args[2] * (-3 / (ztmp * pow(ztmp / (RADIUS_EARTH / NORMFACTOR), 3))) * DIPOLECONST; //mu in [kg.m^2 / s^2.T] = [N.m / T]

	return (F_lor + F_mir) / args[4];
}//returns an acceleration in the parallel direction to the B Field

__device__ double foRungeKuttaCUDA(double* funcArg, int arrayLen, double* LUT)
{	// funcArg requirements: [t_RK = 0, y_0, ...] where t_RK = {0, h/2, h}, initial t_RK should be 0, this func will take care of the rest
	// dy / dt = f(t, y), y(t_0) = y_0
	// remaining funcArg elements are whatever you need in your callback function passed in
	double k1, k2, k3, k4, y_0;
	y_0 = funcArg[1];

	k1 = accel1dCUDA(funcArg, arrayLen, LUT); //k1 = f(t_n, y_n), units of dy / dt

	funcArg[0] = DT / 2;
	funcArg[1] = y_0 + k1 * funcArg[0];
	k2 = accel1dCUDA(funcArg, arrayLen, LUT); //k2 = f(t_n + h/2, y_n + h/2 k1)

	funcArg[1] = y_0 + k2 * funcArg[0];
	k3 = accel1dCUDA(funcArg, arrayLen, LUT); //k3 = f(t_n + h/2, y_n + h/2 k2)

	funcArg[0] = DT;
	funcArg[1] = y_0 + k3 * funcArg[0];
	k4 = accel1dCUDA(funcArg, arrayLen, LUT); //k4 = f(t_n + h, y_n + h k3)
	
	return (k1 + 2 * k2 + 2 * k3 + k4) * DT / 6; //returns delta y, not dy / dt, not total y
}

__global__ void computeKernel(double* v_d, double* mu_d, double* z_d, bool* inSimBool, int* numEscaped, bool elecTF, hiprandStateMRG32k3a* crndStateA, double simtime, double* LUT)
{
	int thdInd = blockIdx.x * blockDim.x + threadIdx.x;
	int nrmGenIdx = (blockIdx.x * 2) + (threadIdx.x % 2);//256 threads per block, 2 random generators per block, 128 threads per RG
	double mass;
	double q;
	double v_sigma;

	if (elecTF)//would have to be changed in the event of multiple ion species
	{
		mass = MASS_ELECTRON;
		q = -1.0;
		v_sigma = sqrt(V_SIGMA_SQ_ELEC);
	}
	else
	{
		mass = MASS_PROTON;
		q = 1.0;
		v_sigma = sqrt(V_SIGMA_SQ_IONS);
	}

#ifdef CUDANORMAL_TEST
	v_d[thdInd] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ));
	mu_d[thdInd] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ)) * 1e-21;

	if (thdInd % 2 == 0)
	{
		z_d[thdInd] = IONSPH_MIN_Z + 0.1;
		v_d[thdInd] = abs(v_d[thdInd]);
	}
	else
	{
		z_d[thdInd] = MAGSPH_MAX_Z - 0.1;
		v_d[thdInd] = -abs(v_d[thdInd]) * sqrt(T_RATIO);
	}
	inSimBool[thdInd] = true;
	return;
#endif

	inSimBool[thdInd] = ((z_d[thdInd] < MAX_Z_SIM) && (z_d[thdInd] > MIN_Z_SIM)); //Makes sure particles are within bounds

	double args[7];

	if (REPLENISH_E_I)
	{
		if (!inSimBool[thdInd])
		{
			inSimBool[thdInd] = true;
			v_d[thdInd] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, v_sigma * VPARACONST);
			numEscaped[thdInd] += 1;
			if (z_d[thdInd] < MIN_Z_SIM)
			{
				z_d[thdInd] = MIN_Z_SIM + 0.01 * (RADIUS_EARTH / NORMFACTOR);
				v_d[thdInd] = abs(v_d[thdInd]);
			}
			else
			{
				z_d[thdInd] = MAX_Z_SIM - 0.01 * (RADIUS_EARTH / NORMFACTOR);
				mu_d[thdInd] *= T_RATIO;
				v_d[thdInd] = -abs(v_d[thdInd]) * sqrt(T_RATIO);
			}
			mu_d[thdInd] = pow(normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, v_sigma), 2) * 0.5 * mass / BFieldatZ(z_d[thdInd], simtime);
		}
	}

	if (inSimBool[thdInd])
	{//args array: [t_RKiter, vz, mu, q, m, pz_0, simtime]
		args[0] = 0.0;
		args[1] = v_d[thdInd];
		args[2] = mu_d[thdInd];
		args[3] = CHARGE_ELEM * q;
		args[4] = mass;
		args[5] = z_d[thdInd];
		args[6] = simtime;

		v_d[thdInd] += foRungeKuttaCUDA(args, 7, LUT);
		z_d[thdInd] += v_d[thdInd] * DT;
	}
}

void Simulation170925::initializeSimulation()
{
	//Allocate memory on GPU for elec/ions variables
	gpuDblMemoryPointers_m.reserve(numberOfParticleTypes_m * numberOfAttributesTracked_m + 1);
	for (int iii = 0; iii < numberOfParticleTypes_m * numberOfAttributesTracked_m + 1; iii++)
	{//[0] = v_e_para, [1] = mu_e_para, [2] = z_e, [3-5] = same attributes for ions, [6] = E Field LUT
		hipMalloc((void **)&gpuDblMemoryPointers_m[iii], DBLARRAY_BYTES);
		if (iii < numberOfParticleTypes_m)
		{
			hipMalloc((void **)&gpuBoolMemoryPointers_m[iii], BOOLARRAY_BYTES); //for inSim bool per particle
			hipMalloc((void **)&gpuIntMemoryPointers_m[iii], INTARRAY_BYTES); //for escaped particle count
			hipMemset(gpuIntMemoryPointers_m[iii], 0, INTARRAY_BYTES); //setting escaped particle count to 0
		}
	}

	//Code to prepare random number generator to produce pseudo-random numbers (for normal dist)
	gpuOtherMemoryPointers_m.reserve(1);
	if (REPLENISH_E_I)
	{
		hiprandStateMRG32k3a* mrgStates_dev;
		long long seed = time(NULL);
		hipMalloc((void **)&mrgStates_dev, 392 * 2 * sizeof(hiprandStateMRG32k3a));
		initCurand <<< 49, 16 >>> (mrgStates_dev, seed); //2 per block, 128 threads per random generator
		gpuOtherMemoryPointers_m[0] = mrgStates_dev;
	}
	else
		gpuOtherMemoryPointers_m[0] = nullptr;

	initialized_m = true;
}

void Simulation170925::copyDataToGPU()
{//copies particle distribution and associated data to GPU in preparation of iterative calculations over the data
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	//copies double arrays associated with particle distribution
	for (int iii = 0; iii < numberOfParticleTypes_m; iii++)
	{
		for (int jjj = 0; jjj < numberOfAttributesTracked_m; jjj++)
			hipMemcpy(gpuDblMemoryPointers_m[iii * numberOfAttributesTracked_m + jjj], particles_m[iii][jjj], DBLARRAY_BYTES, hipMemcpyHostToDevice);

		hipMemcpy(gpuBoolMemoryPointers_m[iii], particlesInSim_m[iii], BOOLARRAY_BYTES, hipMemcpyHostToDevice);
	}

	//copies E field LUT to the GPU
	double LUTtmp[3 * 2951];
	for (int iii = 0; iii < 3; iii++)
	{
		for (int jjj = 0; jjj < 2951; jjj++)
			LUTtmp[iii * 2951 + jjj] = elcFieldLUT_m[iii][jjj];
	}

	hipMemcpy(gpuDblMemoryPointers_m[6], LUTtmp, 3 * 2951 * sizeof(double), hipMemcpyHostToDevice);
	
	copied_m = true;
	
#ifdef LUTCOPY_TEST
	int serializedLUTerr{ 0 };
	int LUTtblerr{ 0 };
	double LUTfromGPU[3 * 2951];
	int tablerow{ -1 };
	hipMemcpy(LUTfromGPU, gpuDblMemoryPointers_m[6], 3 * 2951 * sizeof(double), hipMemcpyDeviceToHost);
	for (int iii = 0; iii < 3 * 2951; iii++)
	{
		if (iii % 2951 == 0)
			tablerow++;
		int tablecol{ iii % 2951 };
		if (LUTfromGPU[iii] != LUTtmp[iii])
			serializedLUTerr++;
		if (LUTfromGPU[iii] != elcFieldLUT_m[tablerow][tablecol])
			LUTtblerr++;
	}
	std::cout << "\n\nSerialized Errors: " << serializedLUTerr << "\nTable Errors: " << LUTtblerr << "\n\n";
#endif
}

void Simulation170925::iterateSimulation(int numberOfIterations)
{//conducts iterative calculations of data previously copied to GPU - runs the data through the computeKernel
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.  You also need to copy data to the GPU with Simulation::copyDataToGPU.\n";
		return;
	}

	if (!copied_m)
	{
		std::cout << "You haven't copied any data to the GPU with Simulation::copyDataToGPU.  Do that first or the GPU has no numbers to work on.\n";
		return;
	}

	double** satelliteGPUDataPtrs[2];
	for (int iii = 0; iii < satellites_m.size(); iii++)
		satelliteGPUDataPtrs[iii] = new double*[3];
	satelliteGPUDataPtrs[0][0] = gpuDblMemoryPointers_m[0];//electrons
	satelliteGPUDataPtrs[0][1] = gpuDblMemoryPointers_m[1];
	satelliteGPUDataPtrs[0][2] = gpuDblMemoryPointers_m[2];
	satelliteGPUDataPtrs[1][0] = gpuDblMemoryPointers_m[3];//ions
	satelliteGPUDataPtrs[1][1] = gpuDblMemoryPointers_m[4];
	satelliteGPUDataPtrs[1][2] = gpuDblMemoryPointers_m[5];
	
	//Make room for 100 measurements
	satelliteData_m.reserve(100);

	//Loop code
	long cudaloopind{ 0 };
	while (cudaloopind < numberOfIterations)
	{
		computeKernel <<< numberOfParticlesPerType_m / BLOCKSIZE, BLOCKSIZE >>> (gpuDblMemoryPointers_m[0], gpuDblMemoryPointers_m[1], gpuDblMemoryPointers_m[2], 
			gpuBoolMemoryPointers_m[0], gpuIntMemoryPointers_m[0], 1, reinterpret_cast<hiprandStateMRG32k3a*>(gpuOtherMemoryPointers_m[0]), simTime_m, gpuDblMemoryPointers_m[6]);
		computeKernel <<< numberOfParticlesPerType_m / BLOCKSIZE, BLOCKSIZE >>> (gpuDblMemoryPointers_m[3], gpuDblMemoryPointers_m[4], gpuDblMemoryPointers_m[5], 
			gpuBoolMemoryPointers_m[1], gpuIntMemoryPointers_m[1], 0, reinterpret_cast<hiprandStateMRG32k3a*>(gpuOtherMemoryPointers_m[0]), simTime_m, gpuDblMemoryPointers_m[6]);
		for (int iii = 0; iii < satellites_m.size(); iii++)
			satellites_m[iii]->iterateDetector(numberOfParticlesPerType_m / BLOCKSIZE, BLOCKSIZE, satelliteGPUDataPtrs[iii % 2]);
		
		cudaloopind++;
		incTime();

		if (cudaloopind % 1000 == 0)
			std::cout << cudaloopind << " / " << numberOfIterations << "  Sim Time: " << simTime_m << "\n";

		if (cudaloopind % 3000 == 0)
		{
			std::vector<std::vector<double*>> tmpcont; //vector of satellites[attributes]
			tmpcont.reserve(satellites_m.size());
			for (int iii = 0; iii < satellites_m.size(); iii++)
			{
				satellites_m[iii]->copyDataToHost();

				std::vector<double*> tmp; //vector of attributes[individual particles (through double*)]
				for (int jjj = 0; jjj < numberOfAttributesTracked_m; jjj++)
				{
					double* dbltmp = new double[NUMPARTICLES];
					double* satDat{ satellites_m[iii]->getDataArrayPointer(jjj) };
					std::copy(&satDat[0], &satDat[NUMPARTICLES - 1], &dbltmp[0]);
					tmp.push_back(dbltmp);
				}

				for (int jjj = 0; jjj < NUMPARTICLES; jjj++)
					tmp[1][jjj] = sqrt(2 * tmp[1][jjj] * BFieldatZ(tmp[2][jjj], simTime_m) / mass_m[iii % 2]);
				tmpcont.push_back(tmp);
			}
			satelliteData_m.push_back(tmpcont);
		}//end if (cudaloop...
	}//end while (cudaloop...
}

void Simulation170925::copyDataToHost()
{//copies data back to host from GPU
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	for (int iii = 0; iii < numberOfParticleTypes_m; iii++)
	{
		for (int jjj = 0; jjj < numberOfAttributesTracked_m; jjj++)
			hipMemcpy(particles_m[iii][jjj], gpuDblMemoryPointers_m[iii * numberOfAttributesTracked_m + jjj], DBLARRAY_BYTES, hipMemcpyDeviceToHost);
		
		hipMemcpy(particlesInSim_m[iii], gpuBoolMemoryPointers_m[iii], BOOLARRAY_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(particlesEscaped_m[iii], gpuIntMemoryPointers_m[iii], INTARRAY_BYTES, hipMemcpyDeviceToHost);
	}

	//not generic but oh well
	for (int iii = 0; iii < numberOfParticlesPerType_m; iii++)
	{
		totalElecEscaped_m += particlesEscaped_m[0][iii];
		totalIonsEscaped_m += particlesEscaped_m[1][iii];
	}

	std::cout << "Electrons escaped: " << totalElecEscaped_m << "\n";
	std::cout << "Ions escaped:      " << totalIonsEscaped_m << "\n";
}

void Simulation170925::freeGPUMemory()
{//used to free the memory on the GPU that's no longer needed
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	//Destroy previously created rn generator
	if (REPLENISH_E_I)
		hipFree(gpuOtherMemoryPointers_m[0]);

	for (int iii = 0; iii < numberOfParticleTypes_m * numberOfAttributesTracked_m + 1; iii++)
	{
		hipFree(gpuDblMemoryPointers_m[iii]);
		if (iii < numberOfParticleTypes_m)
		{
			hipFree(gpuBoolMemoryPointers_m[iii]);
			hipFree(gpuIntMemoryPointers_m[iii]);
		}
	}

	int e_in_sim{ 0 };
	int i_in_sim{ 0 };
	for (int iii = 0; iii < NUMPARTICLES; iii++)
	{
		if (particlesInSim_m[0][iii])
			e_in_sim++;
		if (particlesInSim_m[1][iii])
			i_in_sim++;
	}

	std::cout << "C++: " << e_in_sim << " " << i_in_sim << " " << ((e_in_sim + i_in_sim) * 3) + 4 << "\n";
	hipProfilerStop(); //For profiling the profiler in the CUDA bundle
}