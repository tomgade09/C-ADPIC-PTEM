#include "hip/hip_runtime.h"
//Standard Library includes
#include <string>
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <time.h>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "include\_simulationvariables.h"
#include "include\Simulation170925.h"

//Array Size Variables
const int DBLARRAY_BYTES { NUMPARTICLES * sizeof(double) }; //Global vars - not my favorite solution, but I suppose it works for now
const int BOOLARRAY_BYTES{ NUMPARTICLES * sizeof(bool) };
const int INTARRAY_BYTES { NUMPARTICLES * sizeof(int) };

__host__ __device__ double EFieldatZ(double z, double simtime)
{//E Field in the direction of B (radially outward)
	if ((z > E_RNG_CENTER + E_RNG_DELTA) || (z < E_RNG_CENTER - E_RNG_DELTA))
		return 0.0;
	return CONSTEFIELD;
}

__host__ __device__ double BFieldatZ(double z, double simtime) //this will change in future iterations
{//for now, a simple dipole field
	return DIPOLECONST / pow(z / (RADIUS_EARTH / NORMFACTOR), 3);
}

__global__ void initCurand(hiprandStateMRG32k3a* state, long long seed)
{
	long long id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__device__ double normalGeneratorCUDA(hiprandStateMRG32k3a* state, long long id, double mean, double sigma)
{
	hiprandStateMRG32k3a localState = state[id];
	
	double res = sigma * hiprand_normal_double(&localState) + mean;
	state[id] = localState;

	return res;
}

__device__ double accel1dCUDA(double* args, int len) //made to pass into 1D Fourth Order Runge Kutta code
{//args array: [t_RK, vz, mu, q, m, pz_0, simtime]
	double F_lor, F_mir, ztmp;
	ztmp = args[5] + args[1] * args[0]; //pz_0 + vz * t_RK
	
	//Lorentz force - simply qE - v x B is taken care of by mu - results in kg.m/s^2 - to convert to Re equivalent - divide by Re
	F_lor = args[3] * EFieldatZ(ztmp, args[6]) / NORMFACTOR; //will need to replace E with a function to calculate in more complex models

	//Mirror force
	F_mir = -args[2] * (-3 / (ztmp * pow(ztmp / (RADIUS_EARTH / NORMFACTOR), 3))) * DIPOLECONST; //mu in [kg.m^2 / s^2.T] = [N.m / T]

	return (F_lor + F_mir) / args[4];
}//returns an acceleration in the parallel direction to the B Field

__device__ double foRungeKuttaCUDA(double* funcArg, int arrayLen)
{	// funcArg requirements: [t_RK = 0, y_0, ...] where t_RK = {0, h/2, h}, initial t_RK should be 0, this func will take care of the rest
	// dy / dt = f(t, y), y(t_0) = y_0
	// remaining funcArg elements are whatever you need in your callback function passed in
	double k1, k2, k3, k4, y_0;
	y_0 = funcArg[1];

	k1 = accel1dCUDA(funcArg, arrayLen); //k1 = f(t_n, y_n), units of dy / dt

	funcArg[0] = DT / 2;
	funcArg[1] = y_0 + k1 * funcArg[0];
	k2 = accel1dCUDA(funcArg, arrayLen); //k2 = f(t_n + h/2, y_n + h/2 k1)

	funcArg[1] = y_0 + k2 * funcArg[0];
	k3 = accel1dCUDA(funcArg, arrayLen); //k3 = f(t_n + h/2, y_n + h/2 k2)

	funcArg[0] = DT;
	funcArg[1] = y_0 + k3 * funcArg[0];
	k4 = accel1dCUDA(funcArg, arrayLen); //k4 = f(t_n + h, y_n + h k3)
	
	return (k1 + 2 * k2 + 2 * k3 + k4) * DT / 6; //returns units of y, not dy / dt
}

__global__ void computeKernel(double* v_d, double* mu_d, double* z_d, bool* inSimBool, int* numEscaped, bool elecTF, hiprandStateMRG32k3a* crndStateA, double simtime)
{
	int iii = blockIdx.x * blockDim.x + threadIdx.x;
	int nrmGenIdx = (blockIdx.x * 2) + (threadIdx.x % 2);
	double mass;
	double q;

	if (elecTF)//would have to be changed in the event of multiple ion species
	{
		mass = MASS_ELECTRON;
		q = -1.0;
	}
	else
	{
		mass = MASS_PROTON;
		q = 1.0;
	}

#ifdef CUDANORMAL_TEST
	v_d[iii] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ));
	mu_d[iii] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ)) * 1e-21;

	if (iii % 2 == 0)
	{
		z_d[iii] = IONSPH_MIN_Z + 0.1;
		v_d[iii] = abs(v_d[iii]);
	}
	else
	{
		z_d[iii] = MAGSPH_MAX_Z - 0.1;
		v_d[iii] = -abs(v_d[iii]);
	}
	inSimBool[iii] = true;
	return;
#endif

	inSimBool[iii] = ((z_d[iii] < MAGSPH_MAX_Z) && (z_d[iii] > IONSPH_MIN_Z)); //Makes sure particles are within bounds

	double args[7];

	if (REPLENISH_E_I)
	{
		if (!inSimBool[iii])
		{
			inSimBool[iii] = true;
			v_d[iii] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ) * VPARACONST);
			numEscaped[iii] += 1;
			if (z_d[iii] < IONSPH_MIN_Z)
			{
				z_d[iii] = IONSPH_MIN_Z + 0.01;
				v_d[iii] = abs(v_d[iii]);
			}
			else
			{
				z_d[iii] = MAGSPH_MAX_Z - 0.01;
				v_d[iii] = -abs(v_d[iii]);
			}
			mu_d[iii] = pow(normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ)), 2) * 0.5 * mass / BFieldatZ(z_d[iii], simtime);
		}
	}

	if (inSimBool[iii])
	{//args array: [t_RKiter, vz, mu, q, m, pz_0, simtime]
		args[0] = 0.0;
		args[1] = v_d[iii];
		args[2] = mu_d[iii];
		args[3] = CHARGE_ELEM * q;
		args[4] = mass;
		args[5] = z_d[iii];
		args[6] = simtime;

		v_d[iii] += foRungeKuttaCUDA(args, 7);
		z_d[iii] += v_d[iii] * DT;
	}
}

void Simulation170925::initializeSimulation()
{
	//Generate z values and convert v_perp to mu here
	for (int iii = 0; iii < numberOfParticleTypes_m; iii++)
	{
		for (int jjj = 0; jjj < numberOfParticlesPerType_m; jjj++)
		{
			if (jjj % 2 == 0) //Generate z, every other particle starts at top/bottom of sim respectively
			{
				particles_m[iii][2][jjj] = IONSPH_MIN_Z + 0.01;
				particles_m[iii][0][jjj] = abs(particles_m[iii][0][jjj]);
			}
			else
			{
				particles_m[iii][2][jjj] = MAGSPH_MAX_Z - 0.01;
				particles_m[iii][0][jjj] = -abs(particles_m[iii][0][jjj]);
			}
		}//end for jjj
	}//end for iii
	
	mass_m.reserve(2);
	mass_m[0] = MASS_ELECTRON;
	mass_m[1] = MASS_PROTON;

	convertVPerpToMu();
	
	//Allocate memory on GPU for elec/ions variables
	for (int iii = 0; iii < numberOfParticleTypes_m * numberOfAttributesTracked_m; iii++)
	{//[0] = v_e_para, [1] = mu_e_para, [2] = z_e, [3-5] = same attributes for ions
		hipMalloc((void **)&gpuDblMemoryPointers_m[iii], DBLARRAY_BYTES);
		if (iii < numberOfParticleTypes_m)
		{
			hipMalloc((void **)&gpuBoolMemoryPointers_m[iii], BOOLARRAY_BYTES); //for inSim bool per particle
			hipMalloc((void **)&gpuIntMemoryPointers_m[iii], INTARRAY_BYTES); //for escaped particle count
			hipMemset(gpuIntMemoryPointers_m[iii], 0, INTARRAY_BYTES); //setting escaped particle count to 0
		}
	}

	//Code to prepare random number generator to produce pseudo-random numbers (for normal dist)
	gpuOtherMemoryPointers_m.reserve(1);
	if (REPLENISH_E_I)
	{
		hiprandStateMRG32k3a* mrgStates_dev;
		long long seed = time(NULL);
		hipMalloc((void **)&mrgStates_dev, 392 * 2 * sizeof(hiprandStateMRG32k3a));
		initCurand <<< 49, 16 >>> (mrgStates_dev, seed); //2 per block, 128 threads per random generator
		gpuOtherMemoryPointers_m[0] = mrgStates_dev;
	}
	else
		gpuOtherMemoryPointers_m[0] = nullptr;

#ifdef NO_NORMALIZE_M
	std::string unitstring{ " m" };
#else
	std::string unitstring{ " Re" };
#endif

	//Print things related to simulation characteristics
	std::cout << "Sim between:      " << IONSPH_MIN_Z << " - " << MAGSPH_MAX_Z << unitstring << "\n";
	std::cout << "E Field between:  " << (E_RNG_CENTER - E_RNG_DELTA) << " - " << (E_RNG_CENTER + E_RNG_DELTA) << unitstring << "\n";
	std::cout << "Const E:          " << CONSTEFIELD << " V/m\n\n";
	std::cout << "Particle Number:  " << numberOfParticlesPerType_m << "\n";
	std::cout << "Iteration Number: " << NUMITERATIONS << "\n";
	std::cout << "Replenish lost p: "; (REPLENISH_E_I) ? (std::cout << "True\n\n") : (std::cout << "False\n\n");

	initialized_m = true;
}

void Simulation170925::copyDataToGPU()
{
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	for (int iii = 0; iii < numberOfParticleTypes_m; iii++)
	{
		for (int jjj = 0; jjj < numberOfAttributesTracked_m; jjj++)
		{
			hipMemcpy(gpuDblMemoryPointers_m[iii * numberOfAttributesTracked_m + jjj], particles_m[iii][jjj], DBLARRAY_BYTES, hipMemcpyHostToDevice);
			//std::cout << "Dbls hipMemcpyHostToDevice: " << iii * numberOfAttributesTracked_m + jjj << "  ";
		}
		//std::cout << "\n";
		hipMemcpy(gpuBoolMemoryPointers_m[iii], particlesInSim_m[iii], BOOLARRAY_BYTES, hipMemcpyHostToDevice);
	}
}

void Simulation170925::iterateSimulation(int numberOfIterations)
{
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	long cudaloopind{ 0 };
	//Loop code
	while (cudaloopind < numberOfIterations)
	{//__global__ void computeKernel(double* v_d, double* mu_d, double* z_d, bool* inSimBool, int* numEscaped, bool elecTF, hiprandStateMRG32k3a* crndStateA, double simtime)
		computeKernel <<< numberOfParticlesPerType_m / BLOCKSIZE, BLOCKSIZE >>>(gpuDblMemoryPointers_m[0], gpuDblMemoryPointers_m[1], gpuDblMemoryPointers_m[2], 
			gpuBoolMemoryPointers_m[0], gpuIntMemoryPointers_m[0], 1, reinterpret_cast<hiprandStateMRG32k3a*>(gpuOtherMemoryPointers_m[0]), simTime_m);
		computeKernel <<< numberOfParticlesPerType_m / BLOCKSIZE, BLOCKSIZE >>> (gpuDblMemoryPointers_m[3], gpuDblMemoryPointers_m[4], gpuDblMemoryPointers_m[5], 
			gpuBoolMemoryPointers_m[1], gpuIntMemoryPointers_m[1], 0, reinterpret_cast<hiprandStateMRG32k3a*>(gpuOtherMemoryPointers_m[0]), simTime_m);
		cudaloopind++;
		incTime();

		if (cudaloopind % 1000 == 0)
			std::cout << cudaloopind << " / " << numberOfIterations << "\n";
	}
}

void Simulation170925::copyDataToHost()
{
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	for (int iii = 0; iii < numberOfParticleTypes_m; iii++)
	{
		for (int jjj = 0; jjj < numberOfAttributesTracked_m; jjj++)
		{
			hipMemcpy(particles_m[iii][jjj], gpuDblMemoryPointers_m[iii * numberOfAttributesTracked_m + jjj], DBLARRAY_BYTES, hipMemcpyDeviceToHost);
			//std::cout << "Dbls hipMemcpyDeviceToHost: " << iii * numberOfAttributesTracked_m + jjj << "  ";
		}
		//std::cout << "\n";
		hipMemcpy(particlesInSim_m[iii], gpuBoolMemoryPointers_m[iii], BOOLARRAY_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(particlesEscaped_m[iii], gpuIntMemoryPointers_m[iii], INTARRAY_BYTES, hipMemcpyDeviceToHost);
	}

	//not generic but oh well
	for (int iii = 0; iii < numberOfParticlesPerType_m; iii++)
	{
		totalElecEscaped_m += particlesEscaped_m[0][iii];
		totalIonsEscaped_m += particlesEscaped_m[1][iii];
	}

	std::cout << "Electrons escaped: " << totalElecEscaped_m << "\n";
	std::cout << "Ions escaped:      " << totalIonsEscaped_m << "\n";
}

void Simulation170925::freeGPUMemory()
{	
	if (!initialized_m)
	{
		std::cout << "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.\n";
		return;
	}

	//Destroy previously created rn generator
	if (REPLENISH_E_I)
		hipFree(gpuOtherMemoryPointers_m[0]);

	for (int iii = 0; iii < numberOfParticleTypes_m * numberOfAttributesTracked_m; iii++)
	{
		hipFree(gpuDblMemoryPointers_m[iii]);
		if (iii < numberOfParticleTypes_m)
		{
			hipFree(gpuBoolMemoryPointers_m[iii]);
			hipFree(gpuIntMemoryPointers_m[iii]);
		}
	}

	int e_in_sim{ 0 };
	int i_in_sim{ 0 };
	for (int iii = 0; iii < NUMPARTICLES; iii++)
	{
		if (particlesInSim_m[0][iii])
			e_in_sim++;
		if (particlesInSim_m[1][iii])
			i_in_sim++;
	}

	std::cout << "C++: " << e_in_sim << " " << i_in_sim << " " << ((e_in_sim + i_in_sim) * 3) + 4 << "\n";
	//std::cout << "Sim Characteristics post compute - simTime, dt, # Types, # Particles/Type, # attr tracked: \n" << simTime_m << ", " << dt_m << ", " << numberOfParticleTypes_m;
	//std::cout << ", " << numberOfParticlesPerType_m << ", " << numberOfAttributesTracked_m << "\n";
	hipProfilerStop(); //For profiling
}