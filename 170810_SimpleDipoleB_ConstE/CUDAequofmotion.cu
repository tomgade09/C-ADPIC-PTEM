#include "hip/hip_runtime.h"
//Standard Library includes
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <time.h>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "include\_simulationvariables.h"

__host__ __device__ double EFieldatZ(double z)
{
	if ((z > E_RNG_CENTER + E_RNG_DELTA) || (z < E_RNG_CENTER - E_RNG_DELTA))
		return 0.0;
	return CONSTEFIELD;
}

__host__ __device__ double BFieldatZ(double z) //this will change in future iterations
{//for now, a simple dipole field
	return DIPOLECONST / pow(z, 3);
}

__global__ void initKernel(hiprandStateMRG32k3a* state, long long seed)
{
	long long id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__device__ double normalGeneratorCUDA(hiprandStateMRG32k3a* state, long long id, double mean, double sigma)
{
	hiprandStateMRG32k3a localState = state[id];
	
	double res = sigma * hiprand_normal_double(&localState) + mean;
	state[id] = localState;

	return res;
}

__device__ double accel1dCUDA(double* args, int len) //made to pass into 1D Fourth Order Runge Kutta code
{//args array: [dt, vz, mu, q, m, pz_0]
	double F_lor, F_mir;
	//Lorentz force - simply qE - v x B is taken care of by mu - results in kg.m/s^2 - to convert to Re equivalent - divide by Re
	F_lor = args[3] * EFieldatZ(args[5]) / NORMFACTOR; //will need to replace E with a function to calculate in more complex models

	//Mirror force
	F_mir = -args[2] * (-3 / pow(args[5], 4)) * DIPOLECONST; //have function for gradB based on dipole B field - will need to change later

	return (F_lor + F_mir) / args[4];
}//returns an acceleration in the parallel direction to the B Field

__device__ double foRungeKuttaCUDA(double* funcArg, int arrayLen, double h)
{
	double k1, k2, k3, k4, y_0;
	y_0 = funcArg[1];

	k1 = accel1dCUDA(funcArg, arrayLen); //k1 = f(t_n, y_n), units of dy / dt

	funcArg[0] = h / 2;
	funcArg[1] = y_0 + k1 * funcArg[0];
	k2 = accel1dCUDA(funcArg, arrayLen); //k2 = f(t_n + h/2, y_n + h/2 k1)

	funcArg[1] = y_0 + k2 * funcArg[0];
	k3 = accel1dCUDA(funcArg, arrayLen); //k3 = f(t_n + h/2, y_n + h/2 k2)

	funcArg[0] = h;
	funcArg[1] = y_0 + k3 * funcArg[0];
	k4 = accel1dCUDA(funcArg, arrayLen); //k4 = f(t_n + h, y_n + h k3)
	
	return (k1 + 2 * k2 + 2 * k3 + k4) * h / 6; //returns units of y, not dy / dt
}

__global__ void computeKernel(double* v_d, double* mu_d, double* z_d, bool* inSimBool, bool elecTF, hiprandStateMRG32k3a* crndStateA)
{
	int iii = blockIdx.x * blockDim.x + threadIdx.x;
	int nrmGenIdx = (blockIdx.x * 2) + (threadIdx.x % 2);
	double mass;
	double q;

	if (elecTF)
	{
		mass = MASS_ELECTRON;
		q = -1.0;
	}
	else
	{
		mass = MASS_PROTON;
		q = 1.0;
	}

#ifdef CUDANORMAL_TEST
	v_d[iii] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ));
	mu_d[iii] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ)) * 1e-21;
	
	if (iii % 2 == 0)
	{
		z_d[iii] = IONSPH_MIN_Z + 0.1;
		v_d[iii] = abs(v_d[iii]);
	}
	else
	{
		z_d[iii] = MAGSPH_MAX_Z - 0.1;
		v_d[iii] = -abs(v_d[iii]);
	}
	inSimBool[iii] = true;
	return;
#endif

	inSimBool[iii] = ((z_d[iii] < MAGSPH_MAX_Z) && (z_d[iii] > IONSPH_MIN_Z)); //Makes sure particles are within bounds

	double args[6];

	if (REPLENISH_E_I)
	{
		if (!inSimBool[iii])
		{
			inSimBool[iii] = true;
			v_d[iii] = normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ) * VPARACONST);
			if (z_d[iii] < IONSPH_MIN_Z)
			{
				z_d[iii] = IONSPH_MIN_Z + 0.1;
				v_d[iii] = abs(v_d[iii]);
			}
			else
			{
				z_d[iii] = MAGSPH_MAX_Z - 0.1;
				v_d[iii] = -abs(v_d[iii]);
			}
			mu_d[iii] = pow(normalGeneratorCUDA(crndStateA, nrmGenIdx, V_DIST_MEAN, sqrt(V_SIGMA_SQ)), 2) * 0.5 * mass / BFieldatZ(z_d[iii]);
		}
	}

	if (inSimBool[iii])
	{
		args[0] = 0.0;
		args[1] = v_d[iii];
		args[2] = mu_d[iii];
		args[3] = CHARGE_ELEM * q;
		args[4] = mass;
		args[5] = z_d[iii];

		v_d[iii] += foRungeKuttaCUDA(args, 6, DT);
		z_d[iii] += v_d[iii] * DT;
	}
}

void mainCUDA(double** electrons, double** ions, bool* elec_in_sim_host, bool* ions_in_sim_host)
{
	long cudaloopind{ 0 };
	double* v_e_para_host; double* mu_e_host; double* z_e_host; double* v_i_para_host; double* mu_i_host; double* z_i_host;
	double* v_e_para_dev; double* mu_e_dev; double* z_e_dev; double* v_i_para_dev; double* mu_i_dev; double* z_i_dev;
	bool* elec_in_sim_dev; bool* ions_in_sim_dev;
	v_e_para_host = electrons[0];
	mu_e_host = electrons[1];
	z_e_host = electrons[2];
	v_i_para_host = ions[0];
	mu_i_host = ions[1];
	z_i_host = ions[2];

	const int DBLARRAY_BYTES{ NUMPARTICLES * sizeof(double) };
	const int BOOLARRAY_BYTES{ NUMPARTICLES * sizeof(bool) };

	//allocate memory on GPU
	hipMalloc((void **) &v_e_para_dev, DBLARRAY_BYTES);
	hipMalloc((void **) &mu_e_dev, DBLARRAY_BYTES);
	hipMalloc((void **) &z_e_dev, DBLARRAY_BYTES);
	hipMalloc((void **) &v_i_para_dev, DBLARRAY_BYTES);
	hipMalloc((void **) &mu_i_dev, DBLARRAY_BYTES);
	hipMalloc((void **) &z_i_dev, DBLARRAY_BYTES);
	hipMalloc((void **) &elec_in_sim_dev, BOOLARRAY_BYTES);
	hipMalloc((void **) &ions_in_sim_dev, BOOLARRAY_BYTES);

	//copy memory to device
	hipMemcpy(v_e_para_dev, v_e_para_host, DBLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(mu_e_dev, mu_e_host, DBLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(z_e_dev, z_e_host, DBLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(v_i_para_dev, v_i_para_host, DBLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(mu_i_dev, mu_i_host, DBLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(z_i_dev, z_i_host, DBLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(elec_in_sim_dev, elec_in_sim_host, BOOLARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(ions_in_sim_dev, ions_in_sim_host, BOOLARRAY_BYTES, hipMemcpyHostToDevice);
	
	//Code to prepare random number generator to produce pseudo-random numbers (for normal dist)
	hiprandStateMRG32k3a* mrgStates_dev;
	
	if (REPLENISH_E_I)
	{
		long long seed = time(NULL);
		hipMalloc((void **) &mrgStates_dev, 392 * 2 * sizeof(hiprandStateMRG32k3a));
		initKernel<<< 49, 16 >>>(mrgStates_dev, seed); //2 per block, 128 threads per random generator
	}
	
	//Loop code
	while (cudaloopind < NUMITERATIONS)
	{
		computeKernel<<< NUMPARTICLES / BLOCKSIZE, BLOCKSIZE >>>(v_e_para_dev, mu_e_dev, z_e_dev, elec_in_sim_dev, 1, mrgStates_dev);
		computeKernel<<< NUMPARTICLES / BLOCKSIZE, BLOCKSIZE >>>(v_i_para_dev, mu_i_dev, z_i_dev, ions_in_sim_dev, 0, mrgStates_dev);
		cudaloopind++;
		hipDeviceSynchronize();
		if (cudaloopind % 1000 == 0)
			std::cout << cudaloopind << " / " << NUMITERATIONS << "\n";
	}

	//Destroy previously created rn generator
	if (REPLENISH_E_I)
		hipFree(mrgStates_dev);

	//Copy data back out
	hipMemcpy(v_e_para_host, v_e_para_dev, DBLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(mu_e_host, mu_e_dev, DBLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(z_e_host, z_e_dev, DBLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(v_i_para_host, v_i_para_dev, DBLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(mu_i_host, mu_i_dev, DBLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(z_i_host, z_i_dev, DBLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(elec_in_sim_host, elec_in_sim_dev, BOOLARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(ions_in_sim_host, ions_in_sim_dev, BOOLARRAY_BYTES, hipMemcpyDeviceToHost);

	//Free memory
	hipFree(v_e_para_dev);
	hipFree(mu_e_dev);
	hipFree(z_e_dev);
	hipFree(v_i_para_dev);
	hipFree(mu_i_dev);
	hipFree(z_i_dev);
	hipFree(elec_in_sim_dev);
	hipFree(ions_in_sim_dev);

	hipProfilerStop(); //For profiling
}