#include "hip/hip_runtime.h"
#include "EField\QSPS.h"

#include ""
#include "ErrorHandling\cudaErrorCheck.h"
#include "ErrorHandling\cudaDeviceMacros.h"

__global__ void setupEnvironmentGPU_QSPS(EElem** qsps, double* altMin, double* altMax, double* magnitude, int numRegions)
{
	ZEROTH_THREAD_ONLY("setupEnvironmentGPU_QSPS", (*qsps) = new QSPS(altMin, altMax, magnitude, numRegions)); //this overloaded constructor is only compiled in the case where __CUDA_ARCH__ is defined
}

__global__ void deleteEnvironmentGPU_QSPS(EElem** qsps)
{
	ZEROTH_THREAD_ONLY("deleteEnvironmentGPU_QSPS", delete (*qsps));
}

#ifndef __CUDA_ARCH__ //host code
__host__ const std::vector<double>& QSPS::altMin() const
{
	return altMin_m;
}

__host__ const std::vector<double>& QSPS::altMax() const 
{
	return altMax_m;
}

__host__ const std::vector<double>& QSPS::magnitude() const 
{
	return magnitude_m;
}
#endif

__host__ QSPS::QSPS(std::vector<double> altMin, std::vector<double> altMax, std::vector<double> magnitude) :
	EElem(), numRegions_m{ (int)magnitude.size() }
{
	if (magnitude.size() != altMin.size() || magnitude.size() != altMax.size())
		throw std::invalid_argument("QSPS::QSPS: invalid parameters passed in magnitude, altMin, altMax: resolved vector lengths are not equal");

	#ifndef __CUDA_ARCH__ //host code
	altMin_m = altMin;       //unfortunately this wrapping is necessary
	altMax_m = altMax;       //as the vectors above also have to be wrapped
	magnitude_m = magnitude; //in an ifndef/endif block so this will compile
	modelName_m = "QSPS";
	#endif /* !__CUDA_ARCH__ */

	setupEnvironment();
}

__host__ __device__ QSPS::~QSPS()
{
	#ifndef __CUDA_ARCH__ //host code
	deleteEnvironment();
	#endif /* !__CUDA_ARCH__ */
}

__host__ void QSPS::setupEnvironment()
{
	#ifndef __CUDA_ARCH__ //host code
	CUDA_API_ERRCHK(hipMalloc((void **)&this_d, sizeof(QSPS**))); //malloc for ptr to ptr to GPU QSPS Obj
	CUDA_API_ERRCHK(hipMalloc((void **)&altMin_d, altMin_m.size() * sizeof(double))); //array of altitude min bounds
	CUDA_API_ERRCHK(hipMalloc((void **)&altMax_d, altMax_m.size() * sizeof(double)));
	CUDA_API_ERRCHK(hipMalloc((void **)&magnitude_d, magnitude_m.size() * sizeof(double))); //array of E magnitude between above min/max
	CUDA_API_ERRCHK(hipMemcpy(altMin_d, altMin_m.data(), altMin_m.size() * sizeof(double), hipMemcpyHostToDevice));
	CUDA_API_ERRCHK(hipMemcpy(altMax_d, altMax_m.data(), altMax_m.size() * sizeof(double), hipMemcpyHostToDevice));
	CUDA_API_ERRCHK(hipMemcpy(magnitude_d, magnitude_m.data(), magnitude_m.size() * sizeof(double), hipMemcpyHostToDevice));

	setupEnvironmentGPU_QSPS <<< 1, 1 >>> (this_d, altMin_d, altMax_d, magnitude_d, (int)(magnitude_m.size()));
	CUDA_KERNEL_ERRCHK_WSYNC(); //creates GPU instance of QSPS
	#endif /* !__CUDA_ARCH__ */
}

__host__ void QSPS::deleteEnvironment()
{
	deleteEnvironmentGPU_QSPS <<< 1, 1 >>> (this_d);
	CUDA_KERNEL_ERRCHK_WSYNC();

	CUDA_API_ERRCHK(hipFree(this_d));
	CUDA_API_ERRCHK(hipFree(altMin_d)); //On device
	CUDA_API_ERRCHK(hipFree(altMax_d));
	CUDA_API_ERRCHK(hipFree(magnitude_d));
}

__host__ __device__ double QSPS::getEFieldAtS(const double s, const double t) const
{
	#ifndef __CUDA_ARCH__ //host code
	for (int ind = 0; ind < magnitude_m.size(); ind++)
	{
		if (s >= altMin_m.at(ind) && s <= altMax_m.at(ind))
			return magnitude_m.at(ind);
	}
	#else //device code
	for (int ind = 0; ind < numRegions_m; ind++)
	{
		if (s >= altMin_d[ind] && s <= altMax_d[ind])
			return magnitude_d[ind];
	}
	#endif /* !__CUDA_ARCH__ */

	return 0.0;
}