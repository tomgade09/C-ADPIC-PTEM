#include "hip/hip_runtime.h"
#include "EField/QSPS.h"

#include ""
#include "utils/serializationHelpers.h"
#include "ErrorHandling/cudaErrorCheck.h"
#include "ErrorHandling/cudaDeviceMacros.h"

using namespace utils::fileIO::serialize;

__global__ void setupEnvironmentGPU_QSPS(EElem** qsps, meters* altMin, meters* altMax, double* magnitude, int numRegions)
{
	ZEROTH_THREAD_ONLY("setupEnvironmentGPU_QSPS", (*qsps) = new QSPS(altMin, altMax, magnitude, numRegions)); //this overloaded constructor is only compiled in the case where __CUDA_ARCH__ is defined
}

__global__ void deleteEnvironmentGPU_QSPS(EElem** qsps)
{
	ZEROTH_THREAD_ONLY("deleteEnvironmentGPU_QSPS", delete ((QSPS*)(*qsps)));
}

#ifndef __CUDA_ARCH__ //host code
__host__ const std::vector<meters>& QSPS::altMin() const
{
	return altMin_m;
}

__host__ const std::vector<meters>& QSPS::altMax() const 
{
	return altMax_m;
}

__host__ const std::vector<double>& QSPS::magnitude() const 
{
	return magnitude_m;
}
#endif

__host__ QSPS::QSPS(std::vector<meters> altMin, std::vector<meters> altMax, std::vector<double> magnitude) :
	EElem("QSPS"), numRegions_m{ (int)magnitude.size() }
{
	if (magnitude.size() != altMin.size() || magnitude.size() != altMax.size())
		throw std::invalid_argument("QSPS::QSPS: invalid parameters passed in magnitude, altMin, altMax: resolved vector lengths are not equal");

	#ifndef __CUDA_ARCH__ //host code
	altMin_m = altMin;       //unfortunately this wrapping is necessary
	altMax_m = altMax;       //as the vectors above also have to be wrapped
	magnitude_m = magnitude; //in an ifndef/endif block so this will compile
	modelName_m = "QSPS";
	#endif /* !__CUDA_ARCH__ */

	if (useGPU_m) setupEnvironment();
}

__device__ QSPS::QSPS(meters* altMin, meters* altMax, meters* magnitude, int numRegions) :
	EElem("QSPS"), altMin_d{ altMin }, altMax_d{ altMax }, magnitude_d{ magnitude }, numRegions_m{ numRegions }
{

}

__host__ __device__ QSPS::~QSPS()
{
	#ifndef __CUDA_ARCH__ //host code
	if (useGPU_m) deleteEnvironment();
	#endif /* !__CUDA_ARCH__ */
}

__host__ void QSPS::setupEnvironment()
{
	#ifndef __CUDA_ARCH__ //host code
	CUDA_API_ERRCHK(hipMalloc((void **)&this_d, sizeof(QSPS**))); //malloc for ptr to ptr to GPU QSPS Obj
	CUDA_API_ERRCHK(hipMalloc((void **)&altMin_d, altMin_m.size() * sizeof(double))); //array of altitude min bounds
	CUDA_API_ERRCHK(hipMalloc((void **)&altMax_d, altMax_m.size() * sizeof(double)));
	CUDA_API_ERRCHK(hipMalloc((void **)&magnitude_d, magnitude_m.size() * sizeof(double))); //array of E magnitude between above min/max
	CUDA_API_ERRCHK(hipMemcpy(altMin_d, altMin_m.data(), altMin_m.size() * sizeof(double), hipMemcpyHostToDevice));
	CUDA_API_ERRCHK(hipMemcpy(altMax_d, altMax_m.data(), altMax_m.size() * sizeof(double), hipMemcpyHostToDevice));
	CUDA_API_ERRCHK(hipMemcpy(magnitude_d, magnitude_m.data(), magnitude_m.size() * sizeof(double), hipMemcpyHostToDevice));

	setupEnvironmentGPU_QSPS <<< 1, 1 >>> (this_d, altMin_d, altMax_d, magnitude_d, (int)(magnitude_m.size()));
	CUDA_KERNEL_ERRCHK_WSYNC(); //creates GPU instance of QSPS
	#endif /* !__CUDA_ARCH__ */
}

__host__ void QSPS::deleteEnvironment()
{
	deleteEnvironmentGPU_QSPS <<< 1, 1 >>> (this_d);
	CUDA_KERNEL_ERRCHK_WSYNC();

	CUDA_API_ERRCHK(hipFree(this_d));
	CUDA_API_ERRCHK(hipFree(altMin_d)); //On device
	CUDA_API_ERRCHK(hipFree(altMax_d));
	CUDA_API_ERRCHK(hipFree(magnitude_d));
}

__host__ __device__ Vperm QSPS::getEFieldAtS(const meters s, const seconds t) const
{
	#ifndef __CUDA_ARCH__ //host code
	for (int ind = 0; ind < magnitude_m.size(); ind++)
	{
		if (s >= altMin_m.at(ind) && s <= altMax_m.at(ind))
			return magnitude_m.at(ind);
	}
	#else //device code
	for (int ind = 0; ind < numRegions_m; ind++)
	{
		if (s >= altMin_d[ind] && s <= altMax_d[ind])
			return magnitude_d[ind];
	}
	#endif /* !__CUDA_ARCH__ */

	return 0.0;
}

__host__ void serialize(string serialFolder) const override
{
	string filename{ serialFolder + string("EField_QSPS.ser") };

	if (std::filesystem::exists(filename))
		cerr << "QSPS::serialize: Warning: filename exists: " << filename << " You are overwriting an existing file.";
	
	ofstream out(filename, std::ofstream::binary);
	if (!out) throw invalid_argument("QSPS::serialize: unable to create file: " + filename);
	
	auto writeStrBuf = [&](const stringbuf& sb)
	{
		out.write(sb.str().c_str(), sb.str().length());
	};

	// ======== write data to file ======== //
	out.write(reinterpret_cast<const char*>(this), sizeof(QSPS));
	writeStrBuf(serializeString(string(name_m)));
	writeStrBuf(serializeDoubleVector(altMin_m));
	writeStrBuf(serializeDoubleVector(altMax_m));
	writeStrBuf(serializeDoubleVector(magnitude_m));

	out.close();
}

__host__ void deserialize(string serialFolder, int nameIndex) override
{
	string filename{ serialFolder + string("EField_QSPS" + to_string(nameIndex) + ".ser") };
	
	ifstream in(filename, std::ifstream::binary);
	if (!in) throw invalid_argument("QSPS::deserialize: unable to open file: " + filename);

	QSPS* qsps{ nullptr };
	vector<char> qspschar(sizeof(QSPS));

	in.read(qspschar.data(), sizeof(QSPS));
	dipb = reinterpret_cast<QSPS*>(qspschar.data());
	
	name_m = deserializeStr(in);
	altMin_m = deserializeDoubleVector(in);
	altMax_m = deserializeDoubleVector(in);
	magnitude_m = deserializeDoubleVector(in);

	useGPU_m = qsps->useGPU_m;

	this_d = nullptr;
}