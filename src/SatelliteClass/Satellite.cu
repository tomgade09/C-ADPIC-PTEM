#include "hip/hip_runtime.h"
//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

//Project specific includes
#include "SatelliteClass\Satellite.h"
#include "ErrorHandling\cudaErrorCheck.h"

__global__ void setup2DArray(double* array1D, double** array2D, int cols, int entries);

__global__ void satelliteDetector(double** data_d, double** capture_d, double simtime, double dt, double altitude, bool upward)
{
	int thdInd = blockIdx.x * blockDim.x + threadIdx.x;
	
	double* v_d; double* mu_d; double* z_d; double* simtime_d; double* index_d;
	double* detected_v_d; double* detected_mu_d; double* detected_z_d;
	v_d = data_d[0]; mu_d = data_d[1]; z_d = data_d[2]; simtime_d = capture_d[3]; index_d = capture_d[4];
	detected_v_d = capture_d[0]; detected_mu_d = capture_d[1]; detected_z_d = capture_d[2];

	double z_minus_vdt{ z_d[thdInd] - v_d[thdInd] * dt };
	
	if (simtime == 0) //not sure I fully like this, but it works
		simtime_d[thdInd] = -1.0;

	if (
		(detected_z_d[thdInd] < 1) &&
			( //no detected particle is in the data array at the thread's index already AND
				//detector is facing down and particle crosses altitude in dt
				((!upward) && (z_d[thdInd] > altitude) && (z_minus_vdt < altitude))
					|| //OR
				//detector is facing up and particle crosses altitude in dt
				((upward) && (z_d[thdInd] < altitude) && (z_minus_vdt > altitude))
			)
		)
	{
		detected_v_d[thdInd] = v_d[thdInd];
		detected_mu_d[thdInd] = mu_d[thdInd];
		detected_z_d[thdInd] = z_d[thdInd];
		simtime_d[thdInd] = simtime;
		index_d[thdInd] = static_cast<double>(thdInd);
	}//particle not removed from sim
}

void Satellite::initializeSatelliteOnGPU()
{
	CUDA_API_ERRCHK(hipMalloc((void **)&satCaptureGPU_m, sizeof(double) * (numberOfAttributes_m + 2) * numberOfParticles_m)); //makes room for data of detected particles
	CUDA_API_ERRCHK(hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 2) * numberOfParticles_m)); //sets values to 0
	CUDA_API_ERRCHK(hipMalloc((void **)&dblppGPU_m.at(1), sizeof(double*) * (numberOfAttributes_m + 2)));

	setup2DArray <<< 1, 1 >>> (satCaptureGPU_m, dblppGPU_m.at(1), numberOfAttributes_m + 2, numberOfParticles_m, true);
	CUDA_KERNEL_ERRCHK_WSYNC();
}

void Satellite::iterateDetector(double simtime, double dt, int blockSize)
{
	if (numberOfParticles_m % blockSize != 0)
		throw std::invalid_argument ("Satellite::iterateDetector: numberOfParticles is not a whole multiple of blocksize, some particles will not be checked");
	
	satelliteDetector <<< numberOfParticles_m / blockSize, blockSize >>> (dblppGPU_m.at(0), dblppGPU_m.at(1), simtime, dt, altitude_m, upwardFacing_m);
}

void Satellite::copyDataToHost()
{// data_m array: [v_para, mu, z, time, partindex][particle number]
	dataAllocateNewMsmtVector();
	std::vector<std::vector<double>>& mostRecent{ data_m.at(data_m.size() - 1) };

	for (int satattr = 0; satattr < numberOfAttributes_m + 2; satattr++)
		CUDA_API_ERRCHK(hipMemcpy(mostRecent.at(satattr).data(), satCaptureGPU_m + satattr * numberOfParticles_m, sizeof(double) * numberOfParticles_m, hipMemcpyDeviceToHost));
	
	CUDA_API_ERRCHK(hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 2) * numberOfParticles_m)); //sets values to 0

	dataReady_m = true; //sets to true the first time called
}

void Satellite::freeGPUMemory()
{
	if (!dataOnGPU_m)
		return;

	dataOnGPU_m = false;
	CUDA_API_ERRCHK(hipFree(satCaptureGPU_m));
	CUDA_API_ERRCHK(hipFree(dblppGPU_m.at(1))); //DO NOT FREE dblppGPU_m.at(0) - this is the 2D data array that the sim uses (not the satellite)
}

std::vector<std::vector<double>> Satellite::getConsolidatedData(bool removeZeros)
{
	if (!dataReady_m)
		copyDataToHost();

	std::vector<std::vector<double>> tmp2D;

	for (int attrs = 0; attrs < numberOfAttributes_m + 2; attrs++)
		tmp2D.push_back(std::vector<double>());

	LOOP_OVER_3D_ARRAY(data_m.size(), data_m.at(iii).size(), numberOfParticles_m, \
		if (removeZeros) //iii is msmt iterator, jjj is attribute iterator, kk is particle iterator
		{
			size_t tind{ data_m.at(iii).size() - 1 };
			if (data_m.at(iii).at(tind).at(kk) >= 0.0)
				tmp2D.at(jjj).push_back(data_m.at(iii).at(jjj).at(kk));
		}
		else
			tmp2D.at(jjj).push_back(data_m.at(iii).at(jjj).at(kk));
	)

		return tmp2D;
}