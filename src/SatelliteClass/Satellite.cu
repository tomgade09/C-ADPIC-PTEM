#include "hip/hip_runtime.h"
//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "_simulationvariables.h"
#include "SatelliteClass\Satellite.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error %d at %s:%d\n",EXIT_FAILURE,__FILE__,__LINE__);}} while(0)

__global__ void setupKernel(double* array1D, double** array2D, int cols, int entrs)
{
	if (blockIdx.x * blockDim.x + threadIdx.x != 0)
		return;

	for (int iii = 0; iii < cols; iii++)
		array2D[iii] = &array1D[iii * entrs];
}

__global__ void satelliteDetector(double** data_d, double** capture_d, double simtime, double dt, double altitude, bool upward)
{
	int thdInd = blockIdx.x * blockDim.x + threadIdx.x;
	
	double* v_d; double* mu_d; double* z_d; double* simtime_d; double* index_d;
	double* detected_v_d; double* detected_mu_d; double* detected_z_d;
	v_d = data_d[0]; mu_d = data_d[1]; z_d = data_d[2]; simtime_d = capture_d[3]; index_d = capture_d[4];
	detected_v_d = capture_d[0]; detected_mu_d = capture_d[1]; detected_z_d = capture_d[2];

	double z_minus_vdt{ z_d[thdInd] - v_d[thdInd] * dt };
	
	if (simtime == 0) //not sure I fully like this, but it works
		simtime_d[thdInd] = -1.0;

	if (
		(detected_z_d[thdInd] < 1) &&
			( //no detected particle is in the data array at the thread's index already AND
				//detector is facing down and particle crosses altitude in dt
				((!upward) && (z_d[thdInd] > altitude) && (z_minus_vdt < altitude))
					|| //OR
				//detector is facing up and particle crosses altitude in dt
				((upward) && (z_d[thdInd] < altitude) && (z_minus_vdt > altitude))
			)
		)
	{
		detected_v_d[thdInd] = v_d[thdInd];
		detected_mu_d[thdInd] = mu_d[thdInd];
		detected_z_d[thdInd] = z_d[thdInd];
		simtime_d[thdInd] = simtime;
		index_d[thdInd] = static_cast<double>(thdInd);
	}//particle not removed from sim
}

void Satellite::initializeSatelliteOnGPU()
{
	//dataAllocateNewMsmtVector(); //make room for the first measurement data set

	CUDA_CALL(hipMalloc((void **)&satCaptureGPU_m, sizeof(double) * (numberOfAttributes_m + 2) * numberOfParticles_m)); //makes room for data of detected particles
	CUDA_CALL(hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 2) * numberOfParticles_m)); //sets values to 0
	CUDA_CALL(hipMalloc((void **)&dblppGPU_m[1], sizeof(double*) * numberOfAttributes_m));

	setupKernel <<< 1, 1 >>> (satCaptureGPU_m, dblppGPU_m[1], numberOfAttributes_m + 2, numberOfParticles_m);
}

void Satellite::iterateDetector(int blockSize, double simtime, double dt)
{
	if (numberOfParticles_m % blockSize != 0)
		std::cout << "Warning: " << name_m << ": Satellite::iterateDetector: numberOfParticles is not a whole multiple of blocksize.  Best case: some particles aren't checked.  Worst case: undefined.\n";
	
	satelliteDetector <<< numberOfParticles_m / blockSize, blockSize >>> (dblppGPU_m.at(0), dblppGPU_m.at(1), simtime, dt, altitude_m, upwardFacing_m);
}

void Satellite::copyDataToHost()
{// data_m array: [v_para, mu, z, time, partindex][particle number]
	dataAllocateNewMsmtVector();
	std::vector<std::vector<double>>& mostRecent{ data_m.at(data_m.size() - 1) };

	for (int satattr = 0; satattr < numberOfAttributes_m + 2; satattr++)
		CUDA_CALL(hipMemcpy(mostRecent.at(satattr).data(), satCaptureGPU_m + satattr * numberOfParticles_m, sizeof(double) * numberOfParticles_m, hipMemcpyDeviceToHost));
	
	CUDA_CALL(hipMemset(satCaptureGPU_m, 0, sizeof(double) * (numberOfAttributes_m + 2) * numberOfParticles_m)); //sets values to 0

	dataReady_m = true; //sets to true the first time called
}

void Satellite::freeGPUMemory()
{
	CUDA_CALL(hipFree(satCaptureGPU_m));
	CUDA_CALL(hipFree(dblppGPU_m.at(1))); //DO NOT FREE dblppGPU_m[0] - this is the 2D data array that the sim uses (not the satellite)
}

std::vector<std::vector<double>> Satellite::getConsolidatedData(bool removeZeros)
{
	if (!dataReady_m)
		copyDataToHost();

	std::vector<std::vector<double>> tmp2D;

	for (int attrs = 0; attrs < numberOfAttributes_m + 2; attrs++)
		tmp2D.push_back(std::vector<double>());

	LOOP_OVER_3D_ARRAY(data_m.size(), data_m.at(iii).size(), numberOfParticles_m, \
		if (removeZeros) //iii is msmt iterator, jjj is attribute iterator, kk is particle iterator
		{
			size_t tind{ data_m.at(iii).size() - 1 };
			if (data_m.at(iii).at(tind).at(kk) >= 0.0)
				tmp2D.at(jjj).push_back(data_m.at(iii).at(jjj).at(kk));
		}
		else
			tmp2D.at(jjj).push_back(data_m.at(iii).at(jjj).at(kk));
	)

		return tmp2D;
}

/*void Satellite::vectorTest(std::vector<double*>& in)
{
	int wrong{ 0 };
	for (int iii = 0; iii < numberOfAttributes_m; iii++)
		for (int jjj = 0; jjj < numberOfParticles_m; jjj++)
			if (in.at(iii)[jjj] != data_m.at(iii)[jjj]) { wrong++; }

	std::cout << "Wrong: " << wrong << "\n";
}*/