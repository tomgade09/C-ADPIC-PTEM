#include "hip/hip_runtime.h"
//Standard Library includes
#include <string>
#include <cmath>
#include <time.h>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

//Project specific includes
#include "physicalconstants.h"
#include "SimulationClass\Simulation.h"
#include "ErrorHandling\cudaErrorCheck.h"
#include "ErrorHandling\SimFatalException.h"

//CUDA Variables - if you change these, don't forget to change the associated hiprand code/blocks/etc
// For Geforce 960M (author's computer) - maximum 1024 threads per block - try this to see if it results in faster code execution sometime
constexpr int  BLOCKSIZE{ 256 }; //Number of threads per block - this is most efficient at a multiple of 128 (256 seems to work well), although 250 has been used with slightly less performance

//Commonly used values
extern const int SIMCHARSIZE{ 3 * sizeof(double) };

__global__ void vperpMuConvert(double** dataToConvert, BField** bfield, double mass, double* time, bool vperpToMu)
{//dataToConvert[0] = vpara, [1] = vperp, [2] = s
	unsigned int thdInd{ blockIdx.x * blockDim.x + threadIdx.x };
	double B_s{ (*bfield)->getBFieldAtS(dataToConvert[2][thdInd], ((time != nullptr) ? time[thdInd] : 0.0)) };
	
	if (dataToConvert[1][thdInd] != 0.0)
	{
		if (vperpToMu)
			dataToConvert[1][thdInd] = 0.5 * mass * dataToConvert[1][thdInd] * dataToConvert[1][thdInd] / B_s;
		else
			dataToConvert[1][thdInd] = sqrt(2 * dataToConvert[1][thdInd] * B_s / mass);
	}
}

__device__ double accel1dCUDA(const double vs_RK, const double t_RK, const double* args, BField** bfield, EField** efield) //made to pass into 1D Fourth Order Runge Kutta code
{//args array: [ps_0, mu, q, m, simtime]
	double F_lor, F_mir, stmp;
	stmp = args[0] + vs_RK * t_RK; //ps_0 + vs_RK * t_RK
	
	//Mirror force
	F_mir = -args[1] * (*bfield)->getGradBAtS(stmp, t_RK + args[4]); //-mu * gradB(pos, runge-kutta time + simtime)
	
	//Lorentz force - simply qE - v x B is taken care of by mu - results in kg.m/s^2 - to convert to Re equivalent - divide by Re
	F_lor = args[2] * (*efield)->getEFieldAtS(stmp, t_RK + args[4]); //q * EFieldatS
	
	return (F_lor + F_mir) / args[3];
}//returns an acceleration in the parallel direction to the B Field

__device__ double foRungeKuttaCUDA(const double y_0, const double h, const double* funcArg, BField** bfield, EField** efield)
{	// funcArg requirements: [t_RK = 0, y_0, ...] where t_RK = {0, h/2, h}, initial t_RK should be 0, this func will take care of the rest
	// dy / dt = f(t, y), y(t_0) = y_0
	// remaining funcArg elements are whatever you need in your callback function passed in
	// args array: [t_RKiter, vs, mu, q, m, ps_0, simtime, dt, omega E, const E]
	double k1, k2, k3, k4; double y{ y_0 }; double t_RK{ 0.0 };

	k1 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k1 = f(t_n, y_n), units of dy / dt
	
	t_RK = h / 2;
	y = y_0 + k1 * t_RK;
	k2 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k2 = f(t_n + h/2, y_n + h/2 k1)

	y = y_0 + k2 * t_RK;
	k3 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k3 = f(t_n + h/2, y_n + h/2 k2)

	t_RK = h;
	y = y_0 + k3 * t_RK;
	k4 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k4 = f(t_n + h, y_n + h k3)

	return (k1 + 2 * k2 + 2 * k3 + k4) * h / 6; //returns delta y, not dy / dt, not total y
}

__global__ void computeKernel(double** currData_d, BField** bfield, EField** efield,
	const double simtime, const double dt, const double mass, const double charge, const double simMin, const double simMax, bool* simDone, const int iter, const int dstep_abort)
{
	unsigned int thdInd{ blockIdx.x * blockDim.x + threadIdx.x };

	double* v_d; double* mu_d; double* s_d; //double* t_incident_d;
	v_d = currData_d[0]; mu_d = currData_d[1]; s_d = currData_d[2]; //t_incident_d = currData_d[3]; //to be implemented

	//if (iter % dstep_abort == 0 && *simDone && t_incident_d[thdInd] > simtime) { *simDone = false; }

	if (s_d[thdInd] < simMin * 0.999) //out of sim to the bottom
		return;
	else if (s_d[thdInd] > simMax * 1.001) //out of sim to the top
		return;
	//else if (t_incident_d[thdInd] > simtime) //particle hasn't "entered the sim" yet
		//return;

	if (iter % dstep_abort == 0 && *simDone) { *simDone = false; }

	//args array: [ps_0, mu, q, m, simtime]
	const double args[]{ s_d[thdInd], mu_d[thdInd], charge, mass, simtime };

	v_d[thdInd] += foRungeKuttaCUDA(v_d[thdInd], dt, args, bfield, efield);
	s_d[thdInd] += v_d[thdInd] * dt;
}


//Simulation member functions
void Simulation::initializeSimulation()
{
	if (BFieldModel_m == nullptr)
		throw SimFatalException("Simulation::initializeSimulation: no Magnetic Field model specified", __FILE__, __LINE__);
	if (particles_m.size() == 0)
		throw SimFatalException("Simulation::initializeSimulation: no particles in simulation, sim cannot be initialized without particles", __FILE__, __LINE__);

	if (EFieldModel_m == nullptr) //make sure an EField (even if empty) exists
		EFieldModel_m = std::make_unique<EField>(); EFieldModel_d = EFieldModel_m->getPtrGPU();

	if (tempSats_m.size() > 0)
	{ LOOP_OVER_1D_ARRAY(tempSats_m.size(), createSatellite(tempSats_m.at(iii).get())); } //create satellites
	else
		std::cerr << "Simulation::initializeSimulation: warning: no satellites created" << std::endl;

	//save particle and satellite names to disk
	std::string parts;
	for (int part = 0; part < particles_m.size(); part++)
	{
		parts += particles_m.at(part)->name();
		if (part != particles_m.size() - 1) { parts += ","; }
	}
	FILE_RDWR_EXCEP_CHECK(fileIO::writeTxtFile(parts, saveRootDir_m + "/_chars/Particles.txt"));

	std::string sats;
	for (int sat = 0; sat < satellites_m.size(); sat++)
	{
		sats += satellites_m.at(sat)->satellite->name();
		if (sat != satellites_m.size() - 1) { sats += ","; }
	}
	FILE_RDWR_EXCEP_CHECK(fileIO::writeTxtFile(sats, saveRootDir_m + "/_chars/Satellites.txt"));


	initialized_m = true;
}

void Simulation::iterateSimulation(int numberOfIterations, int itersBtwCouts)
{//conducts iterative calculations of data previously copied to GPU - runs the data through the computeKernel
	if (!initialized_m)
		throw SimFatalException("Simulation::iterateSimulation: sim not initialized with initializeSimulation()", __FILE__, __LINE__);
	
	printSimAttributes(numberOfIterations, itersBtwCouts);
	
	logFile_m->createTimeStruct("Start Iterate " + std::to_string(numberOfIterations)); //index 1
	logFile_m->writeLogFileEntry("Simulation::iterateSimulation: Start Iteration of Sim:  " + std::to_string(numberOfIterations));

	//Copy data to device
	LOOP_OVER_1D_ARRAY(particles_m.size(), particles_m.at(iii)->copyDataToGPU());
	
	//convert particle vperp data to mu
	for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		vperpMuConvert <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), BFieldModel_d, (*part)->mass(), nullptr, true);
	
	//Setup on GPU variable that checks to see if any threads still have a particle in sim and if not, end iterations early
	bool* simDone_d{ nullptr };
	CUDA_API_ERRCHK(hipMalloc((void**)&simDone_d, sizeof(bool)));

	//Loop code
	long cudaloopind{ 0 };
	while (cudaloopind < numberOfIterations)
	{	
		if (cudaloopind % itersBtwCouts == 0) { CUDA_API_ERRCHK(hipMemset(simDone_d, true, sizeof(bool))); } //if it's going to be checked in tnis iter (every itersBtwCouts iterations), set to true

		for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		{
			computeKernel <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), BFieldModel_d, EFieldModel_d,
				simTime_m, dt_m, (*part)->mass(), (*part)->charge(), simMin_m, simMax_m, simDone_d, cudaloopind, itersBtwCouts); //kernel will set boolean to false if at least one particle is still in sim
		}

		CUDA_KERNEL_ERRCHK_WSYNC_WABORT(); //side effect: hipDeviceSynchronize() needed for computeKernel to function properly, which this macro provides

		for (auto sat = satellites_m.begin(); sat < satellites_m.end(); sat++)
			(*sat)->satellite->iterateDetector(simTime_m, dt_m, BLOCKSIZE);
		
		cudaloopind++;
		incTime();
		if (cudaloopind % itersBtwCouts == 0)
		{
			std::stringstream out;
			out << std::setw(std::to_string(numberOfIterations).size()) << cudaloopind;
			std::cout << out.str() << " / " << numberOfIterations << "  |  Sim Time (s): ";
			out.str(""); out.clear();
			out << std::setw(std::to_string((int)(numberOfIterations) * dt_m).size()) << std::fixed << simTime_m;
			std::cout << out.str() << "  |  Real Time Elapsed (s): ";
			logFile_m->printTimeNowFromLastTS(); //need to add to log file as well?
			std::cout << std::endl;

			bool done{ false };
			CUDA_API_ERRCHK(hipMemcpy(&done, simDone_d, sizeof(bool), hipMemcpyDeviceToHost));
			if (done) { std::cout << "All particles finished early.  Breaking loop." << std::endl; break; }
		}
	}

	CUDA_API_ERRCHK(hipFree(simDone_d));
	
	//Convert particle, satellite mu data to vperp
	for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		vperpMuConvert <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), BFieldModel_d, (*part)->mass(), nullptr, false); //nullptr will need to be changed if B ever becomes time dependent, would require loop to record when it stops tracking the particle

	for (auto sat = satellites_m.begin(); sat < satellites_m.end(); sat++)
		vperpMuConvert <<< (*sat)->particle->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>>  ((*sat)->satellite->get2DDataGPUPtr(), BFieldModel_d, (*sat)->particle->mass(),
		(*sat)->satellite->get1DDataGPUPtr() + 3 * (*sat)->particle->getNumberOfParticles(), false);

	//Copy data back to host
	LOOP_OVER_1D_ARRAY(particles_m.size(), particles_m.at(iii)->copyDataToHost());
	LOOP_OVER_1D_ARRAY(satellites_m.size(), satellites_m.at(iii)->satellite->copyDataToHost());

	saveReady_m = true;

	std::cout << "Total sim time: "; logFile_m->printTimeNowFromFirstTS(); std::cout << " s" << std::endl;

	logFile_m->createTimeStruct("End Iterate " + std::to_string(numberOfIterations)); //index 2
	//logFile_m->writeTimeDiffFromNow(1, "End Iterate " + std::to_string(numberOfIterations));
	logFile_m->writeLogFileEntry("Simulation::iterateSimulation: End Iteration of Sim:  " + std::to_string(numberOfIterations));
}

void Simulation::freeGPUMemory()
{//used to free the memory on the GPU that's no longer needed
	if (!initialized_m)
		throw SimFatalException("Simulation::freeGPUMemory: simulation not initialized with initializeSimulation()", __FILE__, __LINE__);

	if (!dataOnGPU_m) { return; }

	logFile_m->writeLogFileEntry("Simulation::freeGPUMemory: Start free GPU Memory.");

	LOOP_OVER_1D_ARRAY(particles_m.size(), particles_m.at(iii)->freeGPUMemory());
	LOOP_OVER_1D_ARRAY(satellites_m.size(), satellites_m.at(iii)->satellite->freeGPUMemory());

	dataOnGPU_m = false;
	logFile_m->writeLogFileEntry("Simulation::freeGPUMemory: End free GPU Memory.");

	CUDA_API_ERRCHK(hipProfilerStop()); //For profiling with the CUDA bundle
}