#include "BField\BField.h"
#include <cstdlib> //for EXIT_FAILURE
#include <iostream>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error %d at %s:%d\n",EXIT_FAILURE,__FILE__,__LINE__);}} while(0)

__host__ __device__ double getBFieldAtS(double(*fcnPtr)(double*, int), double* args, int count)
{
	return fcnPtr(args, count);
}

void BField::setupCallbacksonGPU()
{
	CUDA_CALL(hipMalloc((void **)&BFieldFcnPtr_d, sizeof(callbackFcn)));
	CUDA_CALL(hipMalloc((void **)&gradBFcnPtr_d,  sizeof(callbackFcn)));

	callSetupCallbacksKernel();
}