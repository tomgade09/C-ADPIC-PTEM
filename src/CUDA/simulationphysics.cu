#include "hip/hip_runtime.h"
//Standard Library includes
#include <string>
#include <iostream>
#include <cmath>
#include <time.h>
#include <vector>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

//Project specific includes
#include "_simulationvariables.h"
#include "SimulationClass\AlfvenLUT.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error %d at %s:%d\n",EXIT_FAILURE,__FILE__,__LINE__);}} while(0)

//CUDA Variables - if you change these, don't forget to change the associated hiprand code/blocks/etc
// For Geforce 960M (author's computer) - maximum 1024 threads per block - try this to see if it results in faster code execution sometime
constexpr int  BLOCKSIZE{ 256 }; //Number of threads per block - this is most efficient at a multiple of 128 (256 seems to work well), although 250 has been used with slightly less performance
constexpr int  NUMRNGSTATES{ 64 * BLOCKSIZE };

//Commonly used values
extern const int SIMCHARSIZE{ 8 * sizeof(double) };

__host__ __device__ double alfvenWaveEbyLUT(double** LUT, double z, double simtime, double omegaE);
__host__ __device__ double alfvenWaveEbyCompute(double z, double simtime);

__host__ __device__ double qspsEatZ(double z, double simtime, double constE)
{
	//if ((z > E_RNG_CENTER + E_RNG_DELTA) || (z < E_RNG_CENTER - E_RNG_DELTA))
		//return 0.0;
	return constE;
}

__host__ __device__ double EFieldatZ(double** LUT, double z, double simtime, double omegaE, double constE, bool qsps, bool alfven)
{
	bool alfLUT { false };
	bool alfCalc{ false };
	
	if (LUT == nullptr && alfven)
		alfCalc = true;
	else if (LUT != nullptr && alfven)
		alfLUT = true;

	return (qsps ? (qspsEatZ(z, simtime, constE)) : (0.0)) + (alfLUT ? (alfvenWaveEbyLUT(LUT, z, simtime, omegaE)) : (0.0)) + (alfCalc ? (alfvenWaveEbyCompute(z, simtime)) : (0.0));
}

__host__ __device__ double BFieldatZ(double z, double simtime)
{//for now, a simple dipole field
	if (z == 0)
		return 0.0; //add an error here if this case is true, at some point

	double norm{ RADIUS_EARTH };

	if ((z < RADIUS_EARTH) && (z > 0))
		norm = 1.0;

	return B0ATTHETA / pow(z / norm, 3); //Bz = B0 at theta * (1/rz(in Re))^3
}

__device__ double accel1dCUDA(double* args, int len, double** LUT, bool qsps, bool alfven) //made to pass into 1D Fourth Order Runge Kutta code
{//args array: [t_RKiter, vz, mu, q, m, pz_0, simtime, dt, omega E, const E]
	double F_lor, F_mir, ztmp;
	ztmp = args[5] + args[1] * args[0]; //pz_0 + vz * t_RK
	
	//Lorentz force - simply qE - v x B is taken care of by mu - results in kg.m/s^2 - to convert to Re equivalent - divide by Re
	F_lor = args[3] * EFieldatZ(LUT, ztmp, args[6] + args[0], args[8], args[9], qsps, alfven); //will need to replace E with a function to calculate in more complex models

	//Mirror force
	F_mir = -args[2] * B0ATTHETA * (-3 / (pow(ztmp / RADIUS_EARTH, 4))) / RADIUS_EARTH; //mu in [kg.m^2 / s^2.T] = [N.m / T]

	return (F_lor + F_mir) / args[4];
}//returns an acceleration in the parallel direction to the B Field

__device__ double foRungeKuttaCUDA(double* funcArg, int arrayLen, double** LUT, bool qsps, bool alfven)
{	// funcArg requirements: [t_RK = 0, y_0, ...] where t_RK = {0, h/2, h}, initial t_RK should be 0, this func will take care of the rest
	// dy / dt = f(t, y), y(t_0) = y_0
	// remaining funcArg elements are whatever you need in your callback function passed in
	//args array: [t_RKiter, vz, mu, q, m, pz_0, simtime, dt, omega E, const E]
	double k1, k2, k3, k4, y_0;
	y_0 = funcArg[1];

	k1 = accel1dCUDA(funcArg, arrayLen, LUT, qsps, alfven); //k1 = f(t_n, y_n), units of dy / dt
	
	funcArg[0] = funcArg[7] / 2;
	funcArg[1] = y_0 + k1 * funcArg[0];
	k2 = accel1dCUDA(funcArg, arrayLen, LUT, qsps, alfven); //k2 = f(t_n + h/2, y_n + h/2 k1)

	funcArg[1] = y_0 + k2 * funcArg[0];
	k3 = accel1dCUDA(funcArg, arrayLen, LUT, qsps, alfven); //k3 = f(t_n + h/2, y_n + h/2 k2)

	funcArg[0] = funcArg[7];
	funcArg[1] = y_0 + k3 * funcArg[0];
	k4 = accel1dCUDA(funcArg, arrayLen, LUT, qsps, alfven); //k4 = f(t_n + h, y_n + h k3)

	return (k1 + 2 * k2 + 2 * k3 + k4) * funcArg[7] / 6; //returns delta y, not dy / dt, not total y
}

__global__ void initCurand(hiprandStateMRG32k3a* state, long long seed)
{
	long long id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void setup2DArray(double* array1D, double** array2D, int cols, int entries)
{//run once on only one thread
	if (blockIdx.x * blockDim.x + threadIdx.x != 0)
		return;
	
	for (int iii = 0; iii < cols; iii++)
		array2D[iii] = &array1D[iii * entries];
}

__device__ void ionosphereGenerator(double* v_part, double* mu_part, double* z_part, double* simConsts, double mass, hiprandStateMRG32k3a* rndState)
{//takes pointers to single particle location in attribute arrays (ex: particle 100: ptr to v[100], ptr to mu[100], ptr to z[100], elecTF, ptr to crndStateA[rnd index of thread]
	double2 v_norm; //v_norm.x = v_para; v_norm.y = v_perp
	v_norm = hiprand_normal2_double(rndState); //more efficient to generate two doubles in the one function than run hiprand_normal_double twice according to CUDA docs
	v_norm.x = v_norm.x * sqrt(simConsts[3] * JOULE_PER_EV / mass) + simConsts[5]; //normal dist -> maxwellian
	v_norm.y = v_norm.y * sqrt(simConsts[3] * JOULE_PER_EV / mass) + simConsts[5]; //normal dist -> maxwellian
	///change to Maxwellian E and pitch angle
	*z_part = simConsts[1];
	*v_part = abs(v_norm.x);
	*mu_part = v_norm.y;
}

__device__ void magnetosphereGenerator(double* v_part, double* mu_part, double* z_part, double* simConsts, double mass, hiprandStateMRG32k3a* rndState)
{
	double2 v_norm; //two normal dist values returned to v_norm.x and v_norm.y; v_norm.x = v_para; v_norm.y = v_perp
	v_norm = hiprand_normal2_double(rndState);
	v_norm.x = v_norm.x * sqrt(simConsts[4] * JOULE_PER_EV / mass) + simConsts[5]; //normal dist -> maxwellian
	v_norm.y = v_norm.y * sqrt(simConsts[4] * JOULE_PER_EV / mass) + simConsts[5]; //normal dist -> maxwellian
	///change to Maxwellian E and pitch angle
	*z_part = simConsts[2];
	*v_part = -abs(v_norm.x);
	*mu_part = v_norm.y;
}

__device__ void ionosphereScattering(double* v_part, double* mu_part, double* z_part, double* simConsts, double mass, hiprandStateMRG32k3a* rndState)
{
	//
	//
	//
	//Physics needs to be improved
	ionosphereGenerator(v_part, mu_part, z_part, simConsts, mass, rndState);
}

__global__ void computeKernel(double** currData_d, double** origData_d, double** LUT, double* simConsts, hiprandStateMRG32k3a* crndStateA,
	double simtime, double mass, double charge, long numParts, bool qsps, bool alfven)
{
	unsigned int thdInd{ blockIdx.x * blockDim.x + threadIdx.x };

	double* v_d; double* mu_d; double* z_d;
	double* v_orig; double* vperp_orig; double* z_orig;
	v_d = currData_d[0]; mu_d = currData_d[1]; z_d = currData_d[2];
	v_orig = origData_d[0]; vperp_orig = origData_d[1]; z_orig = origData_d[2];

	if (z_d[thdInd] < 0.001) //if z is zero (or pretty near zero to account for FP error), generate particles - every other starting at bottom/top of sim
	{//previous way to index curandStates: (blockIdx.x * 2) + (threadIdx.x % 2) - this leads to each block accessing two hiprand states - 128 threads call the same state simultaneously and end up with the same values
		if (thdInd < numParts / 2) //need perhaps a better way to determine distribution of ionosphere/magnetosphere particles
			ionosphereGenerator(&v_d[thdInd], &mu_d[thdInd], &z_d[thdInd], simConsts, mass, &crndStateA[(blockIdx.x % (NUMRNGSTATES / BLOCKSIZE)) * blockDim.x + (threadIdx.x)]);
		else
			magnetosphereGenerator(&v_d[thdInd], &mu_d[thdInd], &z_d[thdInd], simConsts, mass, &crndStateA[(blockIdx.x % (NUMRNGSTATES / BLOCKSIZE)) * blockDim.x + (threadIdx.x)]);
		
		v_orig[thdInd] = v_d[thdInd];
		vperp_orig[thdInd] = mu_d[thdInd];
		z_orig[thdInd] = z_d[thdInd];
		mu_d[thdInd] = 0.5 * mass * mu_d[thdInd] * mu_d[thdInd] / BFieldatZ(z_d[thdInd], simtime);
	}
	else if (simtime == 0) //copies data to arrays that track the initial distribution - if data is loaded in, the above block won't be called
	{
		v_orig[thdInd] = v_d[thdInd];
		vperp_orig[thdInd] = mu_d[thdInd];
		z_orig[thdInd] = z_d[thdInd];
		mu_d[thdInd] = 0.5 * mass * mu_d[thdInd] * mu_d[thdInd] / BFieldatZ(z_d[thdInd], simtime);
	}
	else if (z_d[thdInd] < simConsts[1] * 0.999) //out of sim to the bottom, particle has 50% chance of reflecting, 50% chance of new particle
		//ionosphereScattering(&v_d[thdInd], &mu_d[thdInd], &z_d[thdInd], elecTF, &crndStateA[(blockIdx.x * 2) + (threadIdx.x % 2)]);
		return;
	else if (z_d[thdInd] > simConsts[2] * 1.001) //out of sim to the top, particle is lost, new one generated in its place
		//magnetosphereGenerator(&v_d[thdInd], &mu_d[thdInd], &z_d[thdInd], elecTF, &crndStateA[(blockIdx.x * 2) + (threadIdx.x % 2)]);
		return;
	
	//args array: [t_RKiter, vz, mu, q, m, pz_0, simtime, dt, omega E, const E]
	//simConsts: dt, sim min, sim max, t ion, t mag, v mean, omega E Alfven, QSPS const E
	double args[10];
	args[0] = 0.0;
	args[1] = v_d[thdInd];
	args[2] = mu_d[thdInd];
	args[3] = charge;
	args[4] = mass;
	args[5] = z_d[thdInd];
	args[6] = simtime;
	args[7] = simConsts[0];
	args[8] = simConsts[6]; //omega E
	args[9] = simConsts[7]; //QSPS

	v_d[thdInd] += foRungeKuttaCUDA(args, 10, LUT, qsps, alfven);
	z_d[thdInd] += v_d[thdInd] * simConsts[0];
}

void Simulation::initializeSimulation()
{	
	logFile_m.createTimeStruct("Start Sim Init"); //index 1
	logFile_m.writeTimeDiff(0, 1);

	//
	//Check for user error
	if (particleTypes_m.size() == 0)
	{
		logFile_m.writeErrorEntry("Simulation::initializeSimulation", "No particles in sim.  You need to add particles before calling this function.  Returning.", {});
		errorEncountered = true;
		return;
	}

	if (errorEncountered)
		return;
	//Check for user error complete
	//

	//Allocate room in vectors for GPU Memory Pointers
	gpuDblMemoryPointers_m.resize(2 * particleTypes_m.size() + 2); //part 0 curr data, part 1 curr data... part 0 orig data, part 1 orig data... simconsts, LUT
	gpuOtherMemoryPointers_m.resize(2 * particleTypes_m.size() + 2); //part 0 curr 2D, part 1 curr 2D... part 0 orig 2D, part 1 orig 2D... hiprand, LUT 2D
	satelliteData_m.reserve(100); //not resize...Don't know the exact size here so need to use push_back

	//Allocate memory on GPU for elec/ions variables
	for (int ind = 0; ind < 2 * particleTypes_m.size(); ind++) //[0] = e data, [1] = i data, [2] = e orig data, [3] = i orig data
	{
		Particle* partTmp{ particleTypes_m.at(ind % particleTypes_m.size()) };
		size_t memSize{ partTmp->getNumberOfParticles() * partTmp->getNumberOfAttributes() * sizeof(double) };
		
		CUDA_CALL(hipMalloc((void **)&gpuDblMemoryPointers_m.at(ind), memSize));
		CUDA_CALL(hipMemset(gpuDblMemoryPointers_m.at(ind), 0, memSize));
		CUDA_CALL(hipMalloc((void **)&gpuOtherMemoryPointers_m.at(ind), partTmp->getNumberOfAttributes() * sizeof(double*))); //2D array
	}

	if (tempSats_m.size() > 0)
	{
		LOOP_OVER_1D_ARRAY(tempSats_m.size(), createSatellite(tempSats_m.at(iii)->particleInd, tempSats_m.at(iii)->altitude, tempSats_m.at(iii)->upwardFacing, tempSats_m.at(iii)->name););
	}
	else
		logFile_m.writeLogFileEntry("Warning: Simulation::initializeSimulation: No satellites created.  That's odd.");

	//Array of sim characteristics - dt, sim min, sim max, t ion, t mag, v mean, omega E Alfven, QSPS const E
	CUDA_CALL(hipMalloc((void **)&gpuDblMemoryPointers_m.at(2 * particleTypes_m.size()), SIMCHARSIZE));

	//Array of random number generator states
	CUDA_CALL(hipMalloc((void **)&gpuOtherMemoryPointers_m.at(2 * particleTypes_m.size()), NUMRNGSTATES * sizeof(hiprandStateMRG32k3a))); //sizeof(hiprandStateMRG32k3a) is 72 bytes

	//For derived classes to add code
	initializeFollowOn();

	initialized_m = true;
	logFile_m.createTimeStruct("End Sim Init"); //index 2
	logFile_m.writeTimeDiff(1, 2);
}

void Simulation::copyDataToGPU()
{//copies particle distribution and associated data to GPU in preparation of iterative calculations over the data
	logFile_m.writeLogFileEntry("Simulation::copyDataToGPU: Start copy to GPU");

	//
	//Check for user error
	if (!initialized_m)
	{
		logFile_m.writeErrorEntry("Simulation::copyDataToGPU", "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.", {});
		errorEncountered = true;
		return;
	}

	if (errorEncountered)
		return;
	//Check for user error complete
	//

	//Copies initial data of particles to GPU, if loaded
	for (int parts = 0; parts < particleTypes_m.size(); parts++)
	{
		if (particleTypes_m.at(parts)->getInitDataLoaded())
		{
			Particle* tmpPart{ particleTypes_m.at(parts) };
			size_t memSize{ tmpPart->getNumberOfParticles() * sizeof(double) };
			LOOP_OVER_1D_ARRAY(tmpPart->getNumberOfAttributes(), CUDA_CALL(hipMemcpy(gpuDblMemoryPointers_m.at(parts) + tmpPart->getNumberOfParticles() * iii, tmpPart->getCurrData().at(iii).data(), memSize, hipMemcpyHostToDevice));)
		}
	}

	//Copies array of sim characteristics to GPU - dt, sim min, sim max, t ion, t mag, v mean, omega E Alfven, QSPS Const E
	double data[]{ dt_m, simMin_m, simMax_m, ionT_m, magT_m, vmean_m, 0.0, constE_m };
	CUDA_CALL(hipMemcpy(gpuDblMemoryPointers_m.at(2 * particleTypes_m.size()), data, SIMCHARSIZE, hipMemcpyHostToDevice));
	
	for (int iii = 0; iii < 2 * particleTypes_m.size(); iii++)
		setup2DArray <<< 1, 1 >>> (gpuDblMemoryPointers_m.at(iii), reinterpret_cast<double**>(gpuOtherMemoryPointers_m.at(iii)), particleTypes_m.at(iii % particleTypes_m.size())->getNumberOfAttributes(), particleTypes_m.at(iii % particleTypes_m.size())->getNumberOfParticles());
	
	//Prepare hiprand states for random number generation
	long long seed = time(NULL);
	initCurand <<< NUMRNGSTATES / 256, 256 >>> (reinterpret_cast<hiprandStateMRG32k3a*>(gpuOtherMemoryPointers_m.at(2 * particleTypes_m.size())), seed);
	
	//For derived classes to add code
	copyDataToGPUFollowOn();

	copied_m = true;
	
	logFile_m.writeLogFileEntry("Simulation::copyDataToGPU: End copy to GPU");
}

void Simulation::iterateSimulation(int numberOfIterations, int itersBtwCouts)
{//conducts iterative calculations of data previously copied to GPU - runs the data through the computeKernel
	logFile_m.createTimeStruct("Start Iterate " + std::to_string(numberOfIterations)); //index 3
	logFile_m.writeLogFileEntry("Simulation::iterateSimulation: Start Iteration of Sim:  " + std::to_string(numberOfIterations));
	
	//
	//Check for user error
	if (!initialized_m)
	{
		logFile_m.writeErrorEntry("Simulation::iterateSimulation", "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.  You also need to copy data to the GPU with Simulation::copyDataToGPU.", { std::to_string(numberOfIterations) });
		errorEncountered = true;
		return;
	}

	if (!copied_m)
	{
		logFile_m.writeErrorEntry("Simulation::iterateSimulation", "You haven't copied any data to the GPU with Simulation::copyDataToGPU.  Do that first or the GPU has no numbers to work on.", { std::to_string(numberOfIterations) });
		errorEncountered = true;
		return;
	}

	if (errorEncountered)
		return;
	//Check for user error complete
	//

	//For derived classes to add code
	iterateSimulationFollowOnPreLoop();

	size_t numParts{ particleTypes_m.size() };

	//Loop code
	long cudaloopind{ 0 };
	while (cudaloopind < numberOfIterations)
	{	
		for (int parts = 0; parts < particleTypes_m.size(); parts++)
		{
			Particle* tmpPart{ particleTypes_m.at(parts) };

			computeKernel <<< tmpPart->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> (reinterpret_cast<double**>(gpuOtherMemoryPointers_m.at(parts)), //2D array of particle data
				reinterpret_cast<double**>(gpuOtherMemoryPointers_m.at(parts + particleTypes_m.size())), //2D array for original particle data
				reinterpret_cast<double**>(gpuOtherMemoryPointers_m.at(2 * particleTypes_m.size() + 1)), //2D array of LUT data (nullptr if not used)
				gpuDblMemoryPointers_m.at(2 * numParts), //1D array of sim characteristics
				reinterpret_cast<hiprandStateMRG32k3a*>(gpuOtherMemoryPointers_m.at(2 * numParts)), //1D array of hiprand states
				simTime_m, tmpPart->getMass(), tmpPart->getCharge(), tmpPart->getNumberOfParticles(), useQSPS_m, 0);// (useAlfLUT_m || useAlfCal_m)); //other quantities and flags
		}

		for (int sats = 0; sats < satellites_m.size(); sats++)
			satellites_m.at(sats)->satellite->iterateDetector(BLOCKSIZE, simTime_m, dt_m);
		
		cudaloopind++;
		incTime();

		if (cudaloopind % itersBtwCouts == 0)
		{
			std::stringstream out;
			out << std::setw(std::to_string(numberOfIterations).size()) << cudaloopind; //not sure if I like the setw(std::to_string(blah)) solution...
			std::cout << out.str() << " / " << numberOfIterations << "  |  Sim Time (s): ";
			out.str(""); out.clear();
			out << std::setw(std::to_string(static_cast<double>(numberOfIterations) * dt_m).size()) << std::fixed << simTime_m; //not sure if I like the setw(std::to_string(blah)) solution...
			std::cout << out.str() << "  |  Real Time Elapsed (s): ";
			logFile_m.printTimeNowFromLastTS(); //need to add to log file as well?
			std::cout << "\n";
		}

		//if (cudaloopind % 2500 == 0)//need better conditional
			//receiveSatelliteData();

		//For derived classes to add code
		iterateSimulationFollowOnInsideLoop();
	}
	receiveSatelliteData(false);
	std::cout << "\nReceive sat data outside main loop.  Remove after.\n";

	//For derived classes to add code
	iterateSimulationFollowOnPostLoop();

	logFile_m.createTimeStruct("End Iterate " + std::to_string(numberOfIterations)); //index 4
	logFile_m.writeTimeDiffFromNow(3, "End Iterate " + std::to_string(numberOfIterations));
	logFile_m.writeLogFileEntry("Simulation::iterateSimulation: End Iteration of Sim:  " + std::to_string(numberOfIterations));
}

void Simulation::copyDataToHost()
{//copies data back to host from GPU
	logFile_m.writeLogFileEntry("Simulation::copyDataToHost: Copy simulation data from GPU back to host");
	
	//
	//Check for user error
	if (!initialized_m)
	{
		logFile_m.writeErrorEntry("Simulation::copyDataToHost", "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.", {});
		errorEncountered = true;
		return;
	}
	
	if (errorEncountered)
		return;
	//Check for user error complete
	//

	for (int parts = 0; parts < particleTypes_m.size(); parts++)
	{
		Particle* tmpPart{ particleTypes_m.at(parts) };
		size_t memSize{ tmpPart->getNumberOfParticles() * sizeof(double) };
		long numParts{ tmpPart->getNumberOfParticles() };
		LOOP_OVER_1D_ARRAY(tmpPart->getNumberOfAttributes(), CUDA_CALL(hipMemcpy(tmpPart->getCurrData().at(iii).data(), gpuDblMemoryPointers_m.at(parts) + numParts * iii, memSize, hipMemcpyDeviceToHost));)
		LOOP_OVER_1D_ARRAY(tmpPart->getNumberOfAttributes(), CUDA_CALL(hipMemcpy(tmpPart->getOrigData().at(iii).data(), gpuDblMemoryPointers_m.at(parts + particleTypes_m.size()) + numParts * iii, memSize, hipMemcpyDeviceToHost));)
	}

	//For derived classes to add code
	copyDataToHostFollowOn();

	logFile_m.writeLogFileEntry("Simulation::copyDataToHost: Done with copying.");
}

void Simulation::freeGPUMemory()
{//used to free the memory on the GPU that's no longer needed
	logFile_m.writeLogFileEntry("Simulation::freeGPUMemory: Start free GPU Memory.");

	//
	//Check for user error
	if (!initialized_m)
	{
		logFile_m.writeErrorEntry("Simulation::freeGPUMemory", "You haven't initialized the simulation yet with Simulation::initializeSimulation.  Do that first.", {});
		return;
	}
	//Check for user error complete
	//

	LOOP_OVER_1D_ARRAY(gpuDblMemoryPointers_m.size(), CUDA_CALL(hipFree(gpuDblMemoryPointers_m.at(iii)));)
	LOOP_OVER_1D_ARRAY(gpuOtherMemoryPointers_m.size(), CUDA_CALL(hipFree(gpuOtherMemoryPointers_m.at(iii)));)

	//For derived classes to add code
	freeGPUMemoryFollowOn();

	freedGPUMem_m = true;

	logFile_m.writeLogFileEntry("Simulation::freeGPUMemory: End free GPU Memory.");

	CUDA_CALL(hipProfilerStop()); //For profiling the profiler in the CUDA bundle}
}