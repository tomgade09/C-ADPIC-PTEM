#include "hip/hip_runtime.h"
//Standard Library includes
#include <string>
#include <cmath>
#include <sstream>
#include <iomanip>

//CUDA includes
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

//Project specific includes
#include "physicalconstants.h"
#include "utils/loopmacros.h"
#include "Simulation/Simulation.h"
#include "ErrorHandling/cudaErrorCheck.h"
#include "ErrorHandling/SimFatalException.h"

//CUDA Variables - if you change these, don't forget to change the associated hiprand code/blocks/etc
// For Geforce 960M (author's computer) - maximum 1024 threads per block - try this to see if it results in faster code execution sometime
constexpr int  BLOCKSIZE{ 256 }; //Number of threads per block - this is most efficient at a multiple of 128 (256 seems to work well), although 250 has been used with slightly less performance

//Commonly used values
extern const int SIMCHARSIZE{ 3 * sizeof(double) };

namespace physics
{
	__global__ void vperpMuConvert(double** dataToConvert, BField** bfield, double mass, bool vperpToMu, int timeInd = 4)
	{//dataToConvert[0] = vpara, [1] = vperp, [2] = s, [3] = t_incident, [4] = t_escape
		unsigned int thdInd{ blockIdx.x * blockDim.x + threadIdx.x };

		if (dataToConvert[1][thdInd] != 0.0)
		{
			double B_s{ (*bfield)->getBFieldAtS(dataToConvert[2][thdInd], dataToConvert[timeInd][thdInd]) };
			if (vperpToMu)
				dataToConvert[1][thdInd] = 0.5 * mass * dataToConvert[1][thdInd] * dataToConvert[1][thdInd] / B_s;
			else
				dataToConvert[1][thdInd] = sqrt(2 * dataToConvert[1][thdInd] * B_s / mass);
		}
	}

	__host__ void vperpMuConvert(const double vpara, double* vperpOrMu, const double s, const double t_convert, BField* bfield, const double mass, const bool vperpToMu)
	{//dataToConvert[0] = vpara, [1] = vperp, [2] = s, [3] = t_incident, [4] = t_escape
		if (*vperpOrMu != 0.0)
		{
			double B_s{ bfield->getBFieldAtS(s, t_convert) };
			if (vperpToMu)
				*vperpOrMu = 0.5 * mass * (*vperpOrMu) * (*vperpOrMu) / B_s;
			else
				*vperpOrMu = sqrt(2 * (*vperpOrMu) * B_s / mass);
		}
	}

	__device__ __host__ double accel1dCUDA(const double vs_RK, const double t_RK, const double* args, BField** bfield, EField** efield) //made to pass into 1D Fourth Order Runge Kutta code
	{//args array: [s_0, mu, q, m, simtime]
		double F_lor, F_mir, stmp;
		stmp = args[0] + vs_RK * t_RK; //ps_0 + vs_RK * t_RK

		//Mirror force
		F_mir = -args[1] * (*bfield)->getGradBAtS(stmp, t_RK + args[4]); //-mu * gradB(pos, runge-kutta time + simtime)

		//Lorentz force - simply qE - v x B is taken care of by mu - results in kg.m/s^2 - to convert to Re equivalent - divide by Re
		F_lor = args[2] * (*efield)->getEFieldAtS(stmp, t_RK + args[4]); //q * EFieldatS

		return (F_lor + F_mir) / args[3];
	}//returns an acceleration in the parallel direction to the B Field

	__device__ __host__ double foRungeKuttaCUDA(const double y_0, const double h, const double* funcArg, BField** bfield, EField** efield)
	{
		// dy / dt = f(t, y), y(t_0) = y_0
		// funcArgs are whatever you need to pass to the equation
		// args array: [s_0, mu, q, m, simtime]
		double k1, k2, k3, k4; double y{ y_0 }; double t_RK{ 0.0 };
		
		k1 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k1 = f(t_n, y_n), returns units of dy / dt

		t_RK = h / 2;
		y = y_0 + k1 * t_RK;
		k2 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k2 = f(t_n + h/2, y_n + h/2 * k1)

		y = y_0 + k2 * t_RK;
		k3 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k3 = f(t_n + h/2, y_n + h/2 * k2)

		t_RK = h;
		y = y_0 + k3 * t_RK;
		k4 = accel1dCUDA(y, t_RK, funcArg, bfield, efield); //k4 = f(t_n + h, y_n + h k3)

		return (k1 + 2 * k2 + 2 * k3 + k4) * h / 6; //returns delta y, not dy / dt, not total y
	}

	__global__ void simActiveCheck(double** currData_d, bool* simDone)
	{
		//Answers the question: Are there no particles left in the simulation?
		//stores the value to simDone, which is defaulted to true, and flipped to false
		//only if t_escape is less than zero for at least one particle
		//(in that case, the sim is not completely done iterating)
		if (*simDone)
		{
			const double* t_escape_d{ currData_d[4] }; //const double* t_incident_d{ currData_d[3] }; //to be implemented

			unsigned int thdInd{ blockIdx.x * blockDim.x + threadIdx.x };

			if (t_escape_d[thdInd] >= 0.0) //particle has escaped the sim
				return;
			else
				(*simDone) = false;
		}
	}

	__global__ void iterateParticle(double** currData_d, BField** bfield, EField** efield,
		const double simtime, const double dt, const double mass, const double charge, const double simmin, const double simmax)
	{
		unsigned int thdInd{ blockIdx.x * blockDim.x + threadIdx.x };

		double* v_d{ currData_d[0] }; const double* mu_d{ currData_d[1] }; double* s_d{ currData_d[2] }; const double* t_incident_d{ currData_d[3] }; double* t_escape_d{ currData_d[4] };

		if (t_escape_d[thdInd] >= 0.0) //particle has escaped, t_escape is >= 0 iff it has both entered and is outside the sim boundaries
			return;
		else if (t_incident_d[thdInd] > simtime) //particle hasn't "entered the sim" yet
			return;
		else if (s_d[thdInd] < simmin * 0.999) //particle is out of sim to the bottom and t_escape not set yet
		{//eventually build in "fuzzy boundary" - maybe eventually create new particle with initial characteristics on escape
			t_escape_d[thdInd] = simtime;
			return;                      //fuzzyIonosphere(); if (t_escape_d[thdInd] >= 0.0 && t_escape_d[thdInd] < simtime) { return; }
		}
		else if (s_d[thdInd] > simmax * 1.001) //particle is out of sim to the top and t_escape not set yet
		{//maybe eventaully create new particle with initial characteristics on escape
			t_escape_d[thdInd] = simtime;
			return;
		}

		//args array: [ps_0, mu, q, m, simtime]
		const double args[]{ s_d[thdInd], mu_d[thdInd], charge, mass, simtime };

		//foRK (plus v0 in this case) gives v at the next time step (indicated vf in this note):
		//for downgoing (as an example), due to the mirror force, vf will be lower than v0 as the mirror force is acting in the opposite direction as v
		//along the path of the particle, ds, and so s will end up higher if we use ds = (vf * dt) than where it would realistically
		//if we use the ds = (v0 * dt), s will be lower down than where it would end up really (due to the fact that the mirror force acting along ds
		//will slow v down as the particle travels along ds), so I take the average of the two and it seems close enough s = (v0 + (v0 + dv)) / 2 * dt = v0 + dv/2 * dt
		//hence the /2 factor below - FYI, this was checked by the particle's energy (steady state, no E Field) remaining the same throughout the simulation
		double v_orig{ v_d[thdInd] };
		v_d[thdInd] += foRungeKuttaCUDA(v_d[thdInd], dt, args, bfield, efield);
		s_d[thdInd] += (v_d[thdInd] + v_orig) / 2 * dt;
	}

	__host__ void iterateParticle(double* vpara, double* mu, double* s, double* t_incident, double* t_escape, BField* bfield, EField* efield,
		const double simtime, const double dt, const double mass, const double charge, const double simmin, const double simmax)
	{
		if (simtime == 0.0) { *t_escape = -1.0; }
		if (*t_escape >= 0.0) //see above function for description of conditions
			return;
		else if (*t_incident > simtime)
			return;
		else if (*s < simmin * 0.999)
		{
			*t_escape = simtime;
			return;
		}
		else if (*s > simmax * 1.001)
		{
			*t_escape = simtime;
			return;
		}

		const double args[]{ *s, *mu, charge, mass, simtime };

		double v_orig{ *vpara };
		*vpara += foRungeKuttaCUDA(*vpara, dt, args, &bfield, &efield);
		*s += (*vpara + v_orig) / 2 * dt;
	}

	/*
	__device__ void fuzzyIonosphere(double& s_d, const double s_esc_absolute, double& v_d, double& t_escape_d, const double simtime)
	{
		if (v_d > 0.0) { return; } //or do we want upgoing to possibly collide???
		if (someRandomGenerator >/<(=)/== someCondition || s_d <= s_esc_absolute)
			t_escape_d = simtime;
		t_escape_d = simtime;
	}
	*/
}

//Simulation member functions
void Simulation::initializeSimulation()
{
	if (BFieldModel_m == nullptr)
		throw SimFatalException("Simulation::initializeSimulation: no Magnetic Field model specified", __FILE__, __LINE__);
	if (particles_m.size() == 0)
		throw SimFatalException("Simulation::initializeSimulation: no particles in simulation, sim cannot be initialized without particles", __FILE__, __LINE__);

	if (EFieldModel_m == nullptr) //make sure an EField (even if empty) exists
		EFieldModel_m = std::make_unique<EField>();
	
	EFieldModel_d = EFieldModel_m->getPtrGPU();

	if (tempSats_m.size() > 0)
	{ LOOP_OVER_1D_ARRAY(tempSats_m.size(), createSatellite(tempSats_m.at(iii).get())); } //create satellites
	else
		std::cerr << "Simulation::initializeSimulation: warning: no satellites created" << std::endl;

	initialized_m = true;
}

void Simulation::__iterateSimCPU(int numberOfIterations, int checkDoneEvery)
{
	using namespace physics;
	for (auto part = particles_m.begin(); part < particles_m.end(); part++)
	{
		std::vector<std::vector<double>> tmp{ (*part)->data(false) };
		for (int ind = 0; ind < (*part)->getNumberOfParticles(); ind++)
		{//convert vperp to mu in Particle memory
			vperpMuConvert(tmp.at(0).at(ind), &tmp.at(1).at(ind), tmp.at(2).at(ind), tmp.at(4).at(ind), BFieldModel_m.get(), (*part)->mass(), true);
		}
		(*part)->loadDataFromMem(tmp, false);
	}

	std::cout << "\tvpara\tvperp\ts\tE\n";

	long cudaloopind{ 0 };
	while (cudaloopind < numberOfIterations)
	{
		bool done{ true };
		for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		{
			std::vector<std::vector<double>> tmp{ (*part)->data(false) };
			for (int ind = 0; ind < (*part)->getNumberOfParticles(); ind++)
			{
				iterateParticle(&tmp.at(0).at(ind), &tmp.at(1).at(ind), &tmp.at(2).at(ind), &tmp.at(3).at(ind), &tmp.at(4).at(ind),
					BFieldModel_m.get(), EFieldModel_m.get(), simTime_m, dt_m, (*part)->mass(), (*part)->charge(), simMin_m, simMax_m);
				if ((cudaloopind % checkDoneEvery == 0) && done && (tmp.at(4).at(ind) < 0.0))
					done = false;
			}
			double vperp{ tmp.at(1).at(0) };
			vperpMuConvert(tmp.at(0).at(0), &vperp, tmp.at(2).at(0), tmp.at(4).at(0), BFieldModel_m.get(), (*part)->mass(), false);
			if (cudaloopind % checkDoneEvery == 0) { std::cout << tmp.at(0).at(0) << "  " << vperp << "  " << tmp.at(2).at(0) << "  " << tmp.at(1).at(0) << "  "; std::cout << std::setprecision(10) << 0.5 * (*part)->mass() * (tmp.at(0).at(0) * tmp.at(0).at(0) + vperp * vperp) / JOULE_PER_EV << std::setprecision(6) << "\n"; }
			(*part)->loadDataFromMem(tmp, false);
		}

		if (cudaloopind % checkDoneEvery == 0)
		{
			//what else here??
			if (done) { std::cout << "All particles finished early.  Breaking loop." << std::endl; break; }
		}

		incTime();
		cudaloopind++;
	}

	for (auto part = particles_m.begin(); part < particles_m.end(); part++)
	{
		std::vector<std::vector<double>> tmp{ (*part)->data(false) };
		for (int ind = 0; ind < (*part)->getNumberOfParticles(); ind++)
		{//convert mu to vperp in Particle memory
			vperpMuConvert(tmp.at(0).at(ind), &tmp.at(1).at(ind), tmp.at(2).at(ind), tmp.at(4).at(ind), BFieldModel_m.get(), (*part)->mass(), false);
		}
		(*part)->loadDataFromMem(tmp, false);
	}
}

void Simulation::iterateSimulation(int numberOfIterations, int checkDoneEvery)
{//conducts iterative calculations of data previously copied to GPU - runs the data through the computeKernel
	using namespace physics;
	
	if (!initialized_m)
		throw SimFatalException("Simulation::iterateSimulation: sim not initialized with initializeSimulation()", __FILE__, __LINE__);
	
	printSimAttributes(numberOfIterations, checkDoneEvery);
	
	logFile_m->createTimeStruct("Start Iterate " + std::to_string(numberOfIterations));
	logFile_m->writeLogFileEntry("Simulation::iterateSimulation: Start Iteration of Sim:  " + std::to_string(numberOfIterations));

	//Copy data to device
	LOOP_OVER_1D_ARRAY(particles_m.size(), particles_m.at(iii)->copyDataToGPU());
	
	//convert particle vperp data to mu
	for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		vperpMuConvert <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), BFieldModel_d, (*part)->mass(), true);
	
	//Setup on GPU variable that checks to see if any threads still have a particle in sim and if not, end iterations early
	bool* simDone_d{ nullptr };
	CUDA_API_ERRCHK(hipMalloc((void**)&simDone_d, sizeof(bool)));

	//Loop code
	long cudaloopind{ 0 };
	while (cudaloopind < numberOfIterations)
	{	
		if (cudaloopind % checkDoneEvery == 0) { CUDA_API_ERRCHK(hipMemset(simDone_d, true, sizeof(bool))); } //if it's going to be checked in tnis iter (every checkDoneEvery iterations), set to true

		for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		{
			iterateParticle <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), BFieldModel_d, EFieldModel_d,
				simTime_m, dt_m, (*part)->mass(), (*part)->charge(), simMin_m, simMax_m);
			
			//kernel will set boolean to false if at least one particle is still in sim
			if (cudaloopind % checkDoneEvery == 0)
				simActiveCheck <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), simDone_d);
		}

		CUDA_KERNEL_ERRCHK_WSYNC_WABORT(); //side effect: hipDeviceSynchronize() needed for computeKernel to function properly, which this macro provides

		for (auto sat = satPartPairs_m.begin(); sat < satPartPairs_m.end(); sat++)
			(*sat)->satellite->iterateDetector(simTime_m, dt_m, BLOCKSIZE);
		
		if (cudaloopind % checkDoneEvery == 0)
		{
			std::stringstream out;
			out << std::setw(std::to_string(numberOfIterations).size()) << cudaloopind;
			std::cout << out.str() << " / " << numberOfIterations << "  |  Sim Time (s): ";
			out.str(""); out.clear();
			out << std::setw(std::to_string((int)(numberOfIterations) * dt_m).size()) << std::fixed << simTime_m;
			std::cout << out.str() << "  |  Real Time Elapsed (s): ";
			logFile_m->printTimeNowFromLastTS(); //need to add to log file as well?
			std::cout << std::endl;

			bool done{ false };
			CUDA_API_ERRCHK(hipMemcpy(&done, simDone_d, sizeof(bool), hipMemcpyDeviceToHost));
			if (done) { std::cout << "All particles finished early.  Breaking loop." << std::endl; break; }
		}

		incTime();
		cudaloopind++;
	}

	CUDA_API_ERRCHK(hipFree(simDone_d));

	//Convert particle, satellite mu data to vperp
	for (auto part = particles_m.begin(); part < particles_m.end(); part++)
		vperpMuConvert <<< (*part)->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>> ((*part)->getCurrDataGPUPtr(), BFieldModel_d, (*part)->mass(), false); //nullptr will need to be changed if B ever becomes time dependent, would require loop to record when it stops tracking the particle

	for (auto sat = satPartPairs_m.begin(); sat < satPartPairs_m.end(); sat++)
		vperpMuConvert <<< (*sat)->particle->getNumberOfParticles() / BLOCKSIZE, BLOCKSIZE >>>  ((*sat)->satellite->get2DDataGPUPtr(), BFieldModel_d, (*sat)->particle->mass(), false, 3);

	//Copy data back to host
	LOOP_OVER_1D_ARRAY(getNumberOfParticleTypes(), particles_m.at(iii)->copyDataToHost());
	LOOP_OVER_1D_ARRAY(getNumberOfSatellites(), satellite(iii)->copyDataToHost());

	saveReady_m = true;
	saveDataToDisk();
	simTime_m = 0.0;

	std::cout << "Total sim time: "; logFile_m->printTimeNowFromFirstTS(); std::cout << " s" << std::endl;

	logFile_m->createTimeStruct("End Iterate " + std::to_string(numberOfIterations));
	logFile_m->writeLogFileEntry("Simulation::iterateSimulation: End Iteration of Sim:  " + std::to_string(numberOfIterations));
}

void Simulation::freeGPUMemory()
{//used to free the memory on the GPU that's no longer needed
	if (!initialized_m)
		throw SimFatalException("Simulation::freeGPUMemory: simulation not initialized with initializeSimulation()", __FILE__, __LINE__);

	if (!dataOnGPU_m) { return; }

	logFile_m->writeLogFileEntry("Simulation::freeGPUMemory: Start free GPU Memory.");

	LOOP_OVER_1D_ARRAY(getNumberOfParticleTypes(), particles_m.at(iii)->freeGPUMemory());
	LOOP_OVER_1D_ARRAY(getNumberOfSatellites(), satellite(iii)->freeGPUMemory());

	dataOnGPU_m = false;
	logFile_m->writeLogFileEntry("Simulation::freeGPUMemory: End free GPU Memory.");

	CUDA_API_ERRCHK(hipProfilerStop()); //For profiling with the CUDA bundle
}
