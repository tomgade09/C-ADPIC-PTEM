#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#define NL {cout << "\n";}

using std::cout;

int main()
{
	int devCount{ 0 };

	if (hipGetDeviceCount(&devCount) != hipSuccess)
	{
		cout << "Get Device Count Error: " << hipGetErrorName(hipGetLastError()) << "  Exiting.\n";
		return 1;
	}

	for (int devIdx = 0; devIdx < devCount; devIdx++)
	{
		hipDeviceProp_t dev;

		if (hipGetDeviceProperties(&dev, devIdx) != hipSuccess)
		{
			cout << "Get Device Properties Error: Device Index: " << devIdx << "  Error: " << hipGetErrorName(hipGetLastError()) << "  Exiting.\n";
			return 1;
		}

		cout << "================ Device: " << devIdx << " ================"; NL; 
		cout << "\tDevice Name:              " << dev.name; NL;
		cout << "\tTotal Global Mem:         " << dev.totalGlobalMem / 1024 / 1024 / 1024 << " GB"; NL;
		cout << "\tMax Threads Per Block:    " << dev.maxThreadsPerBlock; NL;
		cout << "\tWarp Size:                " << dev.warpSize; NL;
		cout << "\tClock Rate:               " << dev.clockRate / 1024 << " MHz"; NL;
		cout << "\tMemory Clock Rate:        " << dev.memoryClockRate / 1024 << " MHz"; NL;
		cout << "\tMemory Bus Width:         " << dev.memoryBusWidth << " bit"; NL;
		cout << "\tCompute Capability:       " << dev.major << "." << dev.minor; NL;
		NL; NL;
	}

    return 0;
}